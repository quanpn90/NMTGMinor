#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <torch/torch.h>
#include <cmath>
#include <math.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#ifdef OLD_GENERATOR_PATH
#include <ATen/CUDAGeneratorImpl.h>
#else
#include <ATen/cuda/CUDAGeneratorImpl.h>
#endif
#include <hiprand/hiprand_kernel.h>

// includes cublaslt
#include <hipblaslt.h>

// constants for fused bias+relu kernel
#define BIAS_RELU_FW_NTHREADS 128 // forward number of thread per block
#define BIAS_RELU_BW_NTHREADS_X 32 // backward number of thread in feature dim
#define BIAS_RELU_BW_NTHREADS_Y 16 // backward number of thread in batch dim
#define BIAS_RELU_RED_PER_THREAD 16 // backward minimal reduction length per thread

// move to a header later on
#define ILP 4
#define BACKCOEFF M_2_SQRTPI * M_SQRT1_2 * 0.5f
template<typename T>
__host__ __device__ __forceinline__ bool is_aligned(T* p){
  return ((uint64_t)p) % (ILP*sizeof(T)) == 0;
}

template<typename T>
__device__ __forceinline__ void load_store(T* dst, T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}
template<typename T>
__device__ __forceinline__ void load_store(T* dst, volatile T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}
template<typename T>
__device__ __forceinline__ void load_store(volatile T* dst, T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}

// Keep ReLU in float only. When using half, cast to float before calling.
__device__ __inline__ float relu(float a) {
  float retf = max(a, 0.f);
  return (retf);
}


// Keep gelu in float only. When using half, cast to float before calling.
__device__ __inline__ float gelu(float a) {
  float retf = a * normcdff(a);
  return (retf);
}


// Keep gelu in float only. When using half, cast to float before calling.
__device__ __inline__ float gelu_back(float dy, float a) {

  // dy is the gradient w.r.t the gelu output
  float cdf = normcdff(a);
  float pdf = BACKCOEFF * expf(-0.5f * a * a);
  float retf = cdf + a * pdf;

  return (dy * retf);
}




// FP64 Wrapper around cublas GEMMEx
hipblasStatus_t mlp_gemm(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    float alpha,
    const double* A,
    int lda,
    const double* B,
    int ldb,
    const float beta,
    double* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      &alpha,
      A,
      HIP_R_64F,
      lda,
      B,
      HIP_R_64F,
      ldb,
      &beta,
      C,
      HIP_R_64F,
      ldc,
      HIP_R_64F,
      HIPBLAS_GEMM_DEFAULT);
}

// FP32 Wrapper around cublas GEMMEx
hipblasStatus_t mlp_gemm(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    float alpha,
    const float* A,
    int lda,
    const float* B,
    int ldb,
    const float beta,
    float* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      &alpha,
      A,
      HIP_R_32F,
      lda,
      B,
      HIP_R_32F,
      ldb,
      &beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      HIPBLAS_GEMM_DEFAULT);
}

// FP16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t mlp_gemm(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    float alpha,
    const at::Half* A,
    int lda,
    const at::Half* B,
    int ldb,
    float beta,
    at::Half* C,
    int ldc) {

  const half halpha = __float2half_rn(alpha);
  const half hbeta = __float2half_rn(beta);
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      &halpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      &hbeta,
      C,
      HIP_R_16F,
      ldc,
      HIPBLAS_COMPUTE_16F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}


// Bias ADD. Assume input X is [features x batch size], column major.
// Bias is one 'features' long vector, with implicit broadcast.
template <typename T>
__global__ void biasAdd_fprop(T *X, T *b, uint batch_size, uint features) {
  T r_x[ILP];
  T r_b[ILP];
  if(is_aligned(X) && is_aligned(b) && features % ILP ==0) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      int row = tid % (features / ILP);
      load_store(r_x, X, 0 , tid);
      load_store(r_b, b, 0 , row);
      #pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]) + static_cast<float>(r_b[ii]);
        r_x[ii] = bias_sum;
      }
      load_store(X, r_x, tid , 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          int row = tid % features;
          r_x[ii] = X[idx];
          r_b[ii] = b[row];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]) + static_cast<float>(r_b[ii]);
        r_x[ii] = bias_sum;
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          X[idx] = r_x[ii];
        }
      }
    }
  }
}


// Bias ADD + ReLU. Assume input X is [features x batch size], column major.
// Activation support fuesed ReLU. Safe to call in-place.
template <typename T>
__global__ void biasAddDropoutGeLU_fprop(T *X, T *Y, T *b, uint8_t *mask, uint batch_size, uint features, float p,
                                         std::pair<uint64_t, uint64_t> seeds) {
  T r_x[ILP];
  T r_b[ILP];
  T r_y[ILP];
  uint8_t r_m[ILP];
//  uint8_t r_m[ILP];
  float pinv = 1.f/(1.f-p);

  if(is_aligned(X) && is_aligned(b) && is_aligned(Y) && features % ILP ==0) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(
        seeds.first,
        tid,
        seeds.second,
        &state);

    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      int row = tid % (features / ILP);
      load_store(r_x, X, 0 , tid);
      load_store(r_b, b, 0 , row);
      load_store(r_y, Y, 0 , tid);
      load_store(r_m, mask, 0, tid);  // mask has the same size with X

      float4 rand = hiprand_uniform4(&state);
      rand.x = rand.x >= p;
      rand.y = rand.y >= p;
      rand.z = rand.z >= p;
      rand.w = rand.w >= p;

#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]) + static_cast<float>(r_b[ii]);
        r_y[ii] = bias_sum;  // store the mm + bias output
        r_x[ii] = gelu(bias_sum) * (float)(&rand.x)[ii]*pinv;  // gelu * dropout mask
        r_m[ii] = (uint8_t)(&rand.x)[ii];  // store the mask values in buffer
      }
      load_store(X, r_x, tid , 0);
      load_store(mask, r_m, tid , 0);
      load_store(Y, r_y, tid , 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(
        seeds.first,
        tid,
        seeds.second,
        &state);

    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {

      float4 rand = hiprand_uniform4(&state);
      rand.x = rand.x >= p;
      rand.y = rand.y >= p;
      rand.z = rand.z >= p;
      rand.w = rand.w >= p;
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          int row = tid % features;
          r_x[ii] = X[idx];
          r_b[ii] = b[row];
          r_m[ii] = mask[idx];
          r_y[ii] = Y[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]) + static_cast<float>(r_b[ii]);
        r_y[ii] = bias_sum;
        r_x[ii] = gelu(bias_sum)*(float)(&rand.x)[ii]*pinv;
        r_m[ii] = (uint8_t)(&rand.x)[ii];
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          X[idx] = r_x[ii];
          mask[idx] = r_m[ii];
          Y[idx] = r_y[ii];
        }
      }
    }
  }
}

template <typename T>
__global__ void biasAddGeLU_fprop(T *X, T *Y, T *b, uint batch_size, uint features) {
  T r_x[ILP];
  T r_b[ILP];
  T r_y[ILP];

  if(is_aligned(X) && is_aligned(b) && is_aligned(Y) && features % ILP ==0) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      int row = tid % (features / ILP);
      load_store(r_x, X, 0 , tid);
      load_store(r_b, b, 0 , row);
      load_store(r_y, Y, 0 , tid);

#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]) + static_cast<float>(r_b[ii]);
        //r_x[ii] = relu(bias_sum)*(&rand.x)[ii]*pinv;
        r_y[ii] = bias_sum;  // store the mm + bias output
        r_x[ii] = gelu(bias_sum);  // gelu * dropout mask
      }
      load_store(X, r_x, tid , 0);
      load_store(Y, r_y, tid , 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {

#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          int row = tid % features;
          r_x[ii] = X[idx];
          r_b[ii] = b[row];
          r_y[ii] = Y[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]) + static_cast<float>(r_b[ii]);
        r_y[ii] = bias_sum;
        r_x[ii] = gelu(bias_sum);
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          X[idx] = r_x[ii];
          Y[idx] = r_y[ii];
        }
      }
    }
  }
}



// Compute grid size for pointwise backward kernel.
// block_x/y is total elment being handled per block, not number of threads
void get_biasAddRelu_bprop_grid_size(
    int yfeat,
    int batch_size,
    int block_x,
    int block_y,
    int* grid_x,
    int* grid_y) {

  *grid_x = (yfeat + block_x - 1) / block_x;
  // Get number of SMs for efficient reduction.
  int num_SMs = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  // can switch to occupancy calculation. use 4 below now for sm_70
  int max_blocks_y = (num_SMs * 4+(*grid_x)-1) / (*grid_x);
  // block_y should be from minimal work per thread
  int nRedSplits = (batch_size + block_y - 1) / block_y;
  // increase number of elem per thread redcution to not launch more than enough
  // kernel adjust work, so here we just launch max block
  *grid_y = std::min(nRedSplits, max_blocks_y);
  return;
}

// Addition done deterministically via a 2-pass approach. Each CTA writes out partial
// sum, and the last CTA in grid Y dimension accumulates partials serially and writes to result.
template <typename T, int UNROLL_FACTOR>
__global__ void biasAdd_bprop(
    T* dY,
    int features,
    int batch_size,
    volatile float* intermediate,
    int* semaphores,
    T* db) {
  // The feature that this thread is responsible for
  int f = blockIdx.x * blockDim.x + threadIdx.x;

  // Compute the span this thread is responsible for
  // For this block
  int b_chunkSize = (batch_size + gridDim.y - 1) / gridDim.y;
  int b_nStart = blockIdx.y * b_chunkSize;
  int b_nSpan = min(batch_size, b_nStart + b_chunkSize) - b_nStart;
  // For this thread
  int chunkSize = (b_chunkSize + blockDim.y - 1) / blockDim.y;
  int nStart = threadIdx.y * chunkSize + b_nStart;
  int nSpan = min(b_nStart + b_nSpan, nStart + chunkSize) - nStart;

  volatile float* out = intermediate + blockIdx.y * features;

  // Flag to trigger last reduction.
  __shared__ bool isLastBlock;
  // we know block size for now
  __shared__ float smem[BIAS_RELU_BW_NTHREADS_X*BIAS_RELU_BW_NTHREADS_Y];

  // Accumulate db in FP32 always
  float db_local = 0;
  if (f < features) {
    int nidx = 0;
    // Handle non-multiple of UNROLL_FACTOR residue
    for (; nidx < nSpan % UNROLL_FACTOR; nidx++) {
      int64_t row, col, flat_idx;
      row = f;
      col = nStart + nidx;
      flat_idx = col * features + row;
      db_local += (float)dY[flat_idx];
    }

    // Handle meat of work
    for (; (nidx + UNROLL_FACTOR - 1) < nSpan; nidx += UNROLL_FACTOR) {
      int64_t row, col, flat_idx;
      row = f;
      col = nStart + nidx;
      flat_idx = col * features + row;
#pragma unroll 4
      for (int u = 0; u < UNROLL_FACTOR; u++) {
        db_local += (float)dY[flat_idx];
        flat_idx += features;
      }
    }

    // naive block reduction on y-dim
    int linear_idx = threadIdx.y * blockDim.x + threadIdx.x;
    smem[linear_idx] = db_local;
  }
  __syncthreads();
  if (f < features) {
    if(threadIdx.y == 0) {
      for(int yidx = 1; yidx < blockDim.y; yidx++){
        db_local += smem[yidx * blockDim.x + threadIdx.x];
      }

      // block result is in db_local now for all threadIdx.y == 0
      // Write out partial result
      out[f] = db_local;
    }
  }
  __threadfence();
  __syncthreads();

  // Increment semaphore and check if this is the last CTA in the grid_y dimension.
  // Only thread (0,0) calls this
  if (threadIdx.x == 0 && threadIdx.y == 0 && f < features) {
    unsigned int sum_idx;
    sum_idx = atomicAdd(&(semaphores[blockIdx.x]), 1);
    isLastBlock = (sum_idx == (gridDim.y - 1));
  }
  __syncthreads();

  db_local = 0;
  // No block reduction for now, only thread (*,0) do grid reduction
  if (isLastBlock && f < features) {
    if(threadIdx.y == 0) {
      for (int n = 0; n < gridDim.y; n++) {
        int row, col;
        row = f;
        col = n;
        db_local += (float)(intermediate[col * features + row]);
      }
      db[f] = (T)db_local;
    }
  }
}




// ReLU. Assume input X is [features x batch size], column major.
// Safe to call in-place.
template <typename T>
__global__ void Gelu_bprop(T *dY, T* H, T *Y, uint features, uint batch_size, T *dX) {
  T r_dy[ILP];
//  T r_y[ILP];
  T r_h[ILP];
  if(is_aligned(dY) &&
     is_aligned(Y) &&
     is_aligned(H) &&
     is_aligned(dX) &&
     features % ILP ==0) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      load_store(r_dy, dY, 0 , tid);
//      load_store(r_y, Y, 0 , tid);
      load_store(r_h, H, 0 , tid);
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
          r_dy[ii] = gelu_back((float)r_dy[ii], (float)r_h[ii]);
      }
      load_store(dX, r_dy, tid, 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          r_dy[ii] = dY[idx];
//          r_y[ii] = Y[idx];
          r_h[ii] = H[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        r_dy[ii] = gelu_back((float)r_dy[ii], (float)r_h[ii]);
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          dX[idx] = r_dy[ii];
        }
      }
    }
  }
}


// ReLU. Assume input X is [features x batch size], column major.
// Safe to call in-place.
template <typename T>
__global__ void GeluDropout_bprop(T *dY, T* H, T *Y, uint8_t* mask, uint features, uint batch_size, T *dX, float p) {
  T r_dy[ILP];
//  T r_y[ILP];
  T r_h[ILP];
  uint8_t r_m[ILP];
  float pinv = 1.0f / (1.0f - p);
  if(is_aligned(dY) &&
     is_aligned(Y) &&
     is_aligned(H) &&
     is_aligned(dX) &&
     features % ILP ==0) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      load_store(r_dy, dY, 0 , tid);
//      load_store(r_y, Y, 0 , tid);
      load_store(r_h, H, 0 , tid);
      load_store(r_m, mask, 0 , tid);
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
            r_dy[ii] = gelu_back((float)r_dy[ii] * (float)r_m[ii] * pinv, (float)r_h[ii]);
      }
      load_store(dX, r_dy, tid, 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          r_dy[ii] = dY[idx];
//          r_y[ii] = Y[idx];
          r_h[ii] = H[idx];
          r_m[ii] = mask[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        r_dy[ii] = gelu_back( (float)r_dy[ii] * (float)r_m[ii] * pinv, (float)r_h[ii] );
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          dX[idx] = r_dy[ii];
        }
      }
    }
  }
}



// Addition done deterministically via a 2-pass approach. Each CTA writes out partial
// sum, and the last CTA in grid Y dimension accumulates partials serially and writes to result.
template <typename T, int UNROLL_FACTOR>
__global__ void biasAddGeLU_bprop(
    T* Y,
    T* H,
    T* dY,
    int features,
    int batch_size,
    T* dX,
    volatile float* intermediate,
    int* semaphores,
    T* db) {
  // The feature that this thread is responsible for
  int f = blockIdx.x * blockDim.x + threadIdx.x;

  // Compute the span this thread is responsible for
  // For this block
  int b_chunkSize = (batch_size + gridDim.y - 1) / gridDim.y;
  int b_nStart = blockIdx.y * b_chunkSize;
  int b_nSpan = min(batch_size, b_nStart + b_chunkSize) - b_nStart;
  // For this thread
  int chunkSize = (b_chunkSize + blockDim.y - 1) / blockDim.y;
  int nStart = threadIdx.y * chunkSize + b_nStart;
  int nSpan = min(b_nStart + b_nSpan, nStart + chunkSize) - nStart;

  volatile float* out = intermediate + blockIdx.y * features;

  // Flag to trigger last reduction.
  __shared__ bool isLastBlock;
  // we know block size for now
  __shared__ float smem[BIAS_RELU_BW_NTHREADS_X*BIAS_RELU_BW_NTHREADS_Y];

  // Accumulate db in FP32 always
  float db_local = 0;
  if (f < features) {
    int nidx = 0;
    // Handle non-multiple of UNROLL_FACTOR residue
    for (; nidx < nSpan % UNROLL_FACTOR; nidx++) {
      int row, col, flat_idx;
      row = f;
      col = nStart + nidx;
      flat_idx = col * features + row;
//      T y_val = Y[flat_idx];
      T h_val = H[flat_idx];
      T dy_val = dY[flat_idx];
      T dx_val;
      dx_val = gelu_back((float)dy_val, (float)h_val);  // gelu backprop
      dX[flat_idx] = dx_val;
      db_local += (float)dx_val;
    }

    // Handle meat of work
    for (; (nidx + UNROLL_FACTOR - 1) < nSpan; nidx += UNROLL_FACTOR) {
      int row, col, flat_idx;
      row = f;
      col = nStart + nidx;
      flat_idx = col * features + row;
#pragma unroll 4
      for (int u = 0; u < UNROLL_FACTOR; u++) {
//        T y_val = Y[flat_idx];
        T dy_val = dY[flat_idx];
        T h_val = H[flat_idx];
        T dx_val;
        dx_val = gelu_back((float)dy_val, (float)h_val);
        dX[flat_idx] = dx_val;
        db_local += (float)dx_val;
        flat_idx += features;
      }
    }

    // naive block reduction on y-dim
    int linear_idx = threadIdx.y * blockDim.x + threadIdx.x;
    smem[linear_idx] = db_local;
  }
  __syncthreads();
  if (f < features) {
    if(threadIdx.y == 0) {
      for(int yidx = 1; yidx < blockDim.y; yidx++){
        db_local += smem[yidx * blockDim.x + threadIdx.x];
      }

      // block result is in db_local now for all threadIdx.y == 0
      // Write out partial result
      out[f] = db_local;
    }
  }
  __threadfence();
  __syncthreads();

  // Increment semaphore and check if this is the last CTA in the grid_y dimension.
  // Only thread (0,0) calls this
  if (threadIdx.x == 0 && threadIdx.y == 0 && f < features) {
    unsigned int sum_idx;
    sum_idx = atomicAdd(&(semaphores[blockIdx.x]), 1);
    isLastBlock = (sum_idx == (gridDim.y - 1));
  }
  __syncthreads();

  db_local = 0;
  // No block reduction for now, only thread (*,0) do grid reduction
  if (isLastBlock && f < features) {
    if(threadIdx.y == 0) {
      for (int n = 0; n < gridDim.y; n++) {
        int row, col;
        row = f;
        col = n;
        db_local += (float)(intermediate[col * features + row]);
      }
      db[f] = (T)db_local;
    }
  }
}

// Addition done deterministically via a 2-pass approach. Each CTA writes out partial
// sum, and the last CTA in grid Y dimension accumulates partials serially and writes to result.
template <typename T, int UNROLL_FACTOR>
__global__ void biasAddGeLU_bprop_aligned(
    T* Y,
    T* H,
    T* dY,
    int features,
    int batch_size,
    T* dX,
    volatile float* intermediate,
    int* semaphores,
    T* db) {
  // The feature that this thread is responsible for
  int f = blockIdx.x * blockDim.x + threadIdx.x;
//  float pinv = 1.0f / (1.0f - p);

  // Compute the span this thread is responsible for
  // For this block
  int b_chunkSize = (batch_size + gridDim.y - 1) / gridDim.y;
  int b_nStart = blockIdx.y * b_chunkSize;
  int b_nSpan = min(batch_size, b_nStart + b_chunkSize) - b_nStart;
  // For this thread
  int chunkSize = (b_chunkSize + blockDim.y - 1) / blockDim.y;
  int nStart = threadIdx.y * chunkSize + b_nStart;
  int nSpan = min(b_nStart + b_nSpan, nStart + chunkSize) - nStart;

  volatile float* out = intermediate + blockIdx.y * features;

  // Flag to trigger last reduction.
  __shared__ bool isLastBlock;

  // Accumulate db in FP32 always
  float db_local[ILP];
  T r_y[ILP];
  T r_h[ILP];
  T r_dy[ILP];
#pragma unroll
  for(int ii=0;ii<ILP;ii++){
    db_local[ii] = 0.f;
  }

  // f always <= features in this case
  //if (f < features) {
  int nidx = 0;

  // Handle non-multiple of UNROLL_FACTOR residue
  for (; nidx < nSpan % UNROLL_FACTOR; nidx++) {
    int row, col, flat_idx;
    row = f;
    col = nStart + nidx;
    flat_idx = col * features / ILP + row;

    load_store(r_y, Y, 0, flat_idx);
    load_store(r_h, H, 0, flat_idx);
    load_store(r_dy, dY, 0, flat_idx);
#pragma unroll
    for(int ii=0;ii<ILP;ii++){
//      if ((float)r_y[ii] <= 0.f)
//        r_dy[ii] = 0;
//      else {
//        r_dy[ii] = r_dy[ii] * pinv;
//      }
      r_dy[ii] = gelu_back((float)r_dy[ii], (float)r_h[ii]);
      db_local[ii] += (float)r_dy[ii];
    }
    load_store(dX, r_dy, flat_idx, 0);
  }

  // Handle meat of work
  for (; (nidx + UNROLL_FACTOR - 1) < nSpan; nidx += UNROLL_FACTOR) {
    int row, col, flat_idx;
    row = f;
    col = nStart + nidx;
    flat_idx = col * features / ILP + row; // total threads in x == features/ILP
#pragma unroll
    for (int u = 0; u < UNROLL_FACTOR; u++) {
      load_store(r_y, Y, 0, flat_idx);
      load_store(r_h, H, 0, flat_idx);
      load_store(r_dy, dY, 0, flat_idx);
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
//        if ((float)r_y[ii] <= 0.f)
//          r_dy[ii] = 0;
//        else
//          r_dy[ii] = r_dy[ii] * pinv;
        r_dy[ii] = gelu_back((float)r_dy[ii], (float)r_h[ii]);
        db_local[ii] += (float)r_dy[ii];
      }
      load_store(dX, r_dy, flat_idx, 0);
      flat_idx += features/ILP;
    }
  }

  // we know block size for now
  __shared__ float smem[BIAS_RELU_BW_NTHREADS_X*BIAS_RELU_BW_NTHREADS_Y*ILP];
  // naive block reduction on y-dim
  int linear_idx = threadIdx.y * blockDim.x + threadIdx.x;
  float* smem_out = smem + ILP * linear_idx;
#pragma unroll
  for(int ii=0;ii<ILP;ii++){
    smem_out[ii] = db_local[ii]; // reuse local dy buffer
  }
  __syncthreads();
  if(threadIdx.y == 0) {
    for(int yidx = 1; yidx < blockDim.y; yidx++){
      float* smem_in = smem + ILP * (yidx * blockDim.x + threadIdx.x);
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
        db_local[ii] += smem_in[ii]; // reuse local dy buffer
      }
    }

    // block result is in db_local now for all threadIdx.y == 0
    if(gridDim.y == 1) {
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
        r_dy[ii] = db_local[ii]; // reuse local dy buffer
      }
      load_store(db, r_dy, f, 0);
      return;
    }

    // Write out partial result
    load_store(out, db_local, f, 0);
  }
  __threadfence();
  __syncthreads();

  // Increment semaphore and check if this is the last CTA in the grid_y dimension.
  // Only thread (0,0) calls this
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    unsigned int sum_idx;
    sum_idx = atomicAdd(&(semaphores[blockIdx.x]), 1);
    isLastBlock = (sum_idx == (gridDim.y - 1));
  }
  __syncthreads();

#pragma unroll
  for(int ii=0;ii<ILP;ii++){
    db_local[ii] = 0.f;
  }
  float r_db[ILP];

  // No block reduction for now, only thread (*,0) do grid reduction
  if (isLastBlock) {
    if(threadIdx.y == 0){
      for (int n = 0; n < gridDim.y; n++) {
        int row, col;
        row = f;
        col = n;
        load_store(r_db, intermediate, 0, col * features / ILP + row);
#pragma unroll
        for(int ii=0;ii<ILP;ii++){
          db_local[ii] += r_db[ii];
        }
      }
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
        r_dy[ii] = db_local[ii]; // reuse local dy buffer
      }
      load_store(db, r_dy, f, 0);
    }
  }
}


///////////// DROPOUT SILU BACKWARD ///////////////////////////////

// Addition done deterministically via a 2-pass approach. Each CTA writes out partial
// sum, and the last CTA in grid Y dimension accumulates partials serially and writes to result.
template <typename T, int UNROLL_FACTOR>
__global__ void biasAddGeLUDropout_bprop(
    T* Y,
    T* H,
    T* dY,
    uint8_t* mask,
    int features,
    int batch_size,
    T* dX,
    volatile float* intermediate,
    int* semaphores,
    T* db,
    float p) {
  // The feature that this thread is responsible for
  int f = blockIdx.x * blockDim.x + threadIdx.x;
  float pinv = 1.0f / (1.0f - p);

  // Compute the span this thread is responsible for
  // For this block
  int b_chunkSize = (batch_size + gridDim.y - 1) / gridDim.y;
  int b_nStart = blockIdx.y * b_chunkSize;
  int b_nSpan = min(batch_size, b_nStart + b_chunkSize) - b_nStart;
  // For this thread
  int chunkSize = (b_chunkSize + blockDim.y - 1) / blockDim.y;
  int nStart = threadIdx.y * chunkSize + b_nStart;
  int nSpan = min(b_nStart + b_nSpan, nStart + chunkSize) - nStart;

  volatile float* out = intermediate + blockIdx.y * features;

  // Flag to trigger last reduction.
  __shared__ bool isLastBlock;
  // we know block size for now
  __shared__ float smem[BIAS_RELU_BW_NTHREADS_X*BIAS_RELU_BW_NTHREADS_Y];

  // Accumulate db in FP32 always
  float db_local = 0;
  if (f < features) {
    int nidx = 0;
    // Handle non-multiple of UNROLL_FACTOR residue
    for (; nidx < nSpan % UNROLL_FACTOR; nidx++) {
      int row, col, flat_idx;
      row = f;
      col = nStart + nidx;
      flat_idx = col * features + row;
//      T y_val = Y[flat_idx];
      T h_val = H[flat_idx];
      T dy_val = dY[flat_idx];
      uint8_t m_val = mask[flat_idx];
      T dx_val;
      dx_val = gelu_back((float)dy_val * float(m_val) * pinv, (float)h_val)  ;  // gelu backprop

      dX[flat_idx] = dx_val;
      db_local += (float)dx_val;
    }

    // Handle meat of work
    for (; (nidx + UNROLL_FACTOR - 1) < nSpan; nidx += UNROLL_FACTOR) {
      int row, col, flat_idx;
      row = f;
      col = nStart + nidx;
      flat_idx = col * features + row;
#pragma unroll 4
      for (int u = 0; u < UNROLL_FACTOR; u++) {
//        T y_val = Y[flat_idx];
        T dy_val = dY[flat_idx];
        T h_val = H[flat_idx];
        uint8_t m_val = mask[flat_idx];
        T dx_val;
        dx_val = gelu_back((float)dy_val * float(m_val) * pinv, (float)h_val) ;
//        if ((float)y_val > 0.f)
//          dx_val = dy_val * pinv;
//        else
//          dx_val = 0;
        dX[flat_idx] = dx_val;
        db_local += (float)dx_val;
        flat_idx += features;
      }
    }

    // naive block reduction on y-dim
    int linear_idx = threadIdx.y * blockDim.x + threadIdx.x;
    smem[linear_idx] = db_local;
  }
  __syncthreads();
  if (f < features) {
    if(threadIdx.y == 0) {
      for(int yidx = 1; yidx < blockDim.y; yidx++){
        db_local += smem[yidx * blockDim.x + threadIdx.x];
      }

      // block result is in db_local now for all threadIdx.y == 0
      // Write out partial result
      out[f] = db_local;
    }
  }
  __threadfence();
  __syncthreads();

  // Increment semaphore and check if this is the last CTA in the grid_y dimension.
  // Only thread (0,0) calls this
  if (threadIdx.x == 0 && threadIdx.y == 0 && f < features) {
    unsigned int sum_idx;
    sum_idx = atomicAdd(&(semaphores[blockIdx.x]), 1);
    isLastBlock = (sum_idx == (gridDim.y - 1));
  }
  __syncthreads();

  db_local = 0;
  // No block reduction for now, only thread (*,0) do grid reduction
  if (isLastBlock && f < features) {
    if(threadIdx.y == 0) {
      for (int n = 0; n < gridDim.y; n++) {
        int row, col;
        row = f;
        col = n;
        db_local += (float)(intermediate[col * features + row]);
      }
      db[f] = (T)db_local;
    }
  }
}

// Addition done deterministically via a 2-pass approach. Each CTA writes out partial
// sum, and the last CTA in grid Y dimension accumulates partials serially and writes to result.
template <typename T, int UNROLL_FACTOR>
__global__ void biasAddGeLUDropout_bprop_aligned(
    T* Y,
    T* H,
    T* dY,
    uint8_t* mask,
    int features,
    int batch_size,
    T* dX,
    volatile float* intermediate,
    int* semaphores,
    T* db,
    float p) {
  // The feature that this thread is responsible for
  int f = blockIdx.x * blockDim.x + threadIdx.x;
  float pinv = 1.0f / (1.0f - p);

  // Compute the span this thread is responsible for
  // For this block
  int b_chunkSize = (batch_size + gridDim.y - 1) / gridDim.y;
  int b_nStart = blockIdx.y * b_chunkSize;
  int b_nSpan = min(batch_size, b_nStart + b_chunkSize) - b_nStart;
  // For this thread
  int chunkSize = (b_chunkSize + blockDim.y - 1) / blockDim.y;
  int nStart = threadIdx.y * chunkSize + b_nStart;
  int nSpan = min(b_nStart + b_nSpan, nStart + chunkSize) - nStart;

  volatile float* out = intermediate + blockIdx.y * features;

  // Flag to trigger last reduction.
  __shared__ bool isLastBlock;

  // Accumulate db in FP32 always
  float db_local[ILP];
  T r_y[ILP];
  T r_h[ILP];
  T r_dy[ILP];
  uint8_t r_m[ILP];
#pragma unroll
  for(int ii=0;ii<ILP;ii++){
    db_local[ii] = 0.f;
  }

  // f always <= features in this case
  //if (f < features) {
  int nidx = 0;

  // Handle non-multiple of UNROLL_FACTOR residue
  for (; nidx < nSpan % UNROLL_FACTOR; nidx++) {
    int row, col, flat_idx;
    row = f;
    col = nStart + nidx;
    flat_idx = col * features / ILP + row;

    load_store(r_y, Y, 0, flat_idx);
    load_store(r_h, H, 0, flat_idx);
    load_store(r_dy, dY, 0, flat_idx);
    load_store(r_m, mask, 0, flat_idx);
#pragma unroll
    for(int ii=0;ii<ILP;ii++){
//      if ((float)r_y[ii] <= 0.f)
//        r_dy[ii] = 0;
//      else {
//        r_dy[ii] = r_dy[ii] * pinv;
//      }
      r_dy[ii] = gelu_back((float)r_dy[ii] * float(r_m[ii]) * pinv, (float)r_h[ii]) ;
      db_local[ii] += (float)r_dy[ii];
    }
    load_store(dX, r_dy, flat_idx, 0);
  }

  // Handle meat of work
  for (; (nidx + UNROLL_FACTOR - 1) < nSpan; nidx += UNROLL_FACTOR) {
    int row, col, flat_idx;
    row = f;
    col = nStart + nidx;
    flat_idx = col * features / ILP + row; // total threads in x == features/ILP
#pragma unroll
    for (int u = 0; u < UNROLL_FACTOR; u++) {
      load_store(r_y, Y, 0, flat_idx);
      load_store(r_h, H, 0, flat_idx);
      load_store(r_dy, dY, 0, flat_idx);
      load_store(r_m, mask, 0, flat_idx);
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
//        if ((float)r_y[ii] <= 0.f)
//          r_dy[ii] = 0;
//        else
//          r_dy[ii] = r_dy[ii] * pinv;
        r_dy[ii] = gelu_back((float)r_dy[ii] * (float)r_m[ii] * pinv, (float)r_h[ii]) ;
        db_local[ii] += (float)r_dy[ii];
      }
      load_store(dX, r_dy, flat_idx, 0);
      flat_idx += features/ILP;
    }
  }

  // we know block size for now
  __shared__ float smem[BIAS_RELU_BW_NTHREADS_X*BIAS_RELU_BW_NTHREADS_Y*ILP];
  // naive block reduction on y-dim
  int linear_idx = threadIdx.y * blockDim.x + threadIdx.x;
  float* smem_out = smem + ILP * linear_idx;
#pragma unroll
  for(int ii=0;ii<ILP;ii++){
    smem_out[ii] = db_local[ii]; // reuse local dy buffer
  }
  __syncthreads();
  if(threadIdx.y == 0) {
    for(int yidx = 1; yidx < blockDim.y; yidx++){
      float* smem_in = smem + ILP * (yidx * blockDim.x + threadIdx.x);
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
        db_local[ii] += smem_in[ii]; // reuse local dy buffer
      }
    }

    // block result is in db_local now for all threadIdx.y == 0
    if(gridDim.y == 1) {
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
        r_dy[ii] = db_local[ii]; // reuse local dy buffer
      }
      load_store(db, r_dy, f, 0);
      return;
    }

    // Write out partial result
    load_store(out, db_local, f, 0);
  }
  __threadfence();
  __syncthreads();

  // Increment semaphore and check if this is the last CTA in the grid_y dimension.
  // Only thread (0,0) calls this
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    unsigned int sum_idx;
    sum_idx = atomicAdd(&(semaphores[blockIdx.x]), 1);
    isLastBlock = (sum_idx == (gridDim.y - 1));
  }
  __syncthreads();

#pragma unroll
  for(int ii=0;ii<ILP;ii++){
    db_local[ii] = 0.f;
  }
  float r_db[ILP];

  // No block reduction for now, only thread (*,0) do grid reduction
  if (isLastBlock) {
    if(threadIdx.y == 0){
      for (int n = 0; n < gridDim.y; n++) {
        int row, col;
        row = f;
        col = n;
        load_store(r_db, intermediate, 0, col * features / ILP + row);
#pragma unroll
        for(int ii=0;ii<ILP;ii++){
          db_local[ii] += r_db[ii];
        }
      }
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
        r_dy[ii] = db_local[ii]; // reuse local dy buffer
      }
      load_store(db, r_dy, f, 0);
    }
  }
}

// Lists where the num_layers-1 intermediate Y buffers start in reserved space on fprop, starting
// offset 0. The last Y value is, of course, stored in the user provided output buffer.
void get_y_offsets(
    int batch_size,
    int num_layers,
    const int* output_features,
    int* y_start_offsets) {
  y_start_offsets[0] = 0;
  for (int i = 1; i < num_layers; i++) {
    y_start_offsets[i] = y_start_offsets[i - 1] + batch_size * output_features[i - 1];
  }
}

// Returns the size of all fprop activations combined
size_t get_all_activations_size(int64_t batch_size, int num_layers, const int* output_features) {
  size_t acts_size = 0;
  for (int l = 0; l < num_layers; l++) {
    acts_size += output_features[l] * batch_size;
  }
  return acts_size;
}

// Returns the reserved space (in elements) needed for the MLP
size_t get_mlp_reserved_space(int64_t batch_size, int num_layers, const int* output_features) {
  size_t res_space = 0;
  // Need to store output of every intermediate MLP - size equal to output_features[i] * batch_size
  // for all 'i' in [0, num_layers-1)
  for (int l = 0; l < num_layers - 1; l++) {
    res_space += output_features[l] * batch_size;
  }
  return res_space;
}

// Returns the size of all fprop activations combined
// no dropout and activation at the last layer so no need that one
size_t get_mlp_activation_space(int64_t batch_size, int num_layers, const int* output_features) {
  size_t acts_size = 0;
  for (int l = 0; l < num_layers - 1; l++) {
    acts_size += output_features[l] * batch_size;
  }
  return acts_size;
}

#if 0
// Returns the work space (in elements) needed for the MLP bprop.
size_t get_mlp_bp_workspace (int batch_size, int num_layers, const int* output_features) {
    /*
       Workspace is partitioned as
       DY_GEMMs : DX_GEMMs
    */
    size_t work_space = 0;

    // Store each intermediate dY explicitly. Need 2 dYs per MLP layer (one for o/p
    // of biasReLU_bp and one for o/p of dgrad GEMM).
    work_space += 2*get_all_activations_size(batch_size, num_layers, output_features);

    return work_space;
}
#endif

// Scratch space needed for reductions in number of elements
size_t get_reduction_scratch_space(int batch_size, int num_layers, const int* output_features) {
  size_t max_scratch_space = 0;
  // Loop over all layers to see which one needs the max scratch space
  for (int l = 0; l < num_layers; l++) {
    // need to find max(aligned, not_aligned)
    int tmp, res0, res1;

    int block_x = BIAS_RELU_BW_NTHREADS_X;
    int block_y = BIAS_RELU_RED_PER_THREAD * BIAS_RELU_BW_NTHREADS_Y;
    get_biasAddRelu_bprop_grid_size(
      output_features[l], batch_size, block_x, block_y, &tmp, &res0);

    block_x = ILP * BIAS_RELU_BW_NTHREADS_X;
    get_biasAddRelu_bprop_grid_size(
      output_features[l], batch_size, block_x, block_y, &tmp, &res1);

    max_scratch_space = std::max(max_scratch_space, (size_t)(output_features[l] * res0));
    max_scratch_space = std::max(max_scratch_space, (size_t)(output_features[l] * res1));
  }

  return max_scratch_space;
}

// Buffer for semaphores
size_t get_semaphores_size(int num_layers, const int* output_features) {
  // Upper bound on semaphores is one per feature for the layer
  // with the most features.
  int max_features = 0;
  for (int l = 0; l < num_layers; l++) {
    max_features = std::max(max_features, output_features[l]);
  }
  return (size_t)max_features;
}

// Returns the work space (in elements) needed for the MLP bprop.
template <typename T>
size_t get_mlp_bp_workspace_in_bytes(int batch_size, int num_layers, const int* output_features) {
  size_t work_space = 0;

  // Store each intermediate dY explicitly. Need 2 dYs per MLP layer (one for o/p
  // of biasReLU_bp and one for o/p of dgrad GEMM).
  work_space += 2 * get_all_activations_size(batch_size, num_layers, output_features) * sizeof(T);
  work_space +=
      get_reduction_scratch_space(batch_size, num_layers, output_features) * sizeof(float);
  work_space += get_semaphores_size(num_layers, output_features) * sizeof(int);

  return work_space;
}

// Returns pointers to each segment of the workspace
template <typename T>
void partition_mlp_bp_workspace(
    int batch_size,
    int num_layers,
    const int* output_features,
    void* work_space,
    T** dy_gemms,
    T** dx_gemms,
    float** db_scratch,
    int** semaphores) {
  /*
     Workspace is partitioned as
     DY_GEMMs : DX_GEMMs : DB_SCRATCH : SEMAPHORES
  */
  // Start address where dy_gemm tensors are stored
  *dy_gemms = reinterpret_cast<T*>(work_space);
  // Start address where dx_gemm tensors are stored
  *dx_gemms = *dy_gemms + get_all_activations_size(batch_size, num_layers, output_features);
  // Start address where db intermediate tensors are stored
  *db_scratch = reinterpret_cast<float*>(
      *dx_gemms + get_all_activations_size(batch_size, num_layers, output_features));
  // Start address of semaphores
  *semaphores = reinterpret_cast<int*>(
      *db_scratch + get_reduction_scratch_space(batch_size, num_layers, output_features));

  return;
}

// Does a simple MLP fprop (GEMM+bias+ReLU).
// Can handle num_layers number of layers, each with its own shape. Output of layer i is assumed
// to be input of layer i+1. output_features, WPtr and BPtr are arrays of length num_layers, and
// must be in the same order i.e. WPtr[i] and BPtr[i] are respectively the weight and bias of layer
// 'i'.
template <typename T>
int mlp_fp(
    T* X,
    int input_features,
    int batch_size,
    T** WPtr,
    int num_layers,
    int* output_features,
    T** BPtr,
    T* Y,
    T* reserved_space,
    T* reserved_activations,
    uint8_t* reserved_mask,
    float p) {
  auto gen = at::cuda::detail::getDefaultCUDAGenerator();
  T *weight, *input, *output, *hidden, *bias;
  uint8_t *mask,  *reserved_space_m;
  T *reserved_space_x, *reserved_space_y, *reserved_space_a;
  reserved_space_x = NULL;
  reserved_space_a = reserved_activations;
  reserved_space_y = reserved_space;
  reserved_space_m = reserved_mask;

  // Get cublas handle from Pytorch
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  // Get the strea* from cublas handle to reuse for biasReLU kernel.
  hipStream_t stream;
  hipblasGetStream(handle, &stream);
//  int activation = 1;


  for (int layer = 0; layer < num_layers; layer++) {
    weight = WPtr[layer];
    input = (layer == 0) ? X : reserved_space_x;
    output = (layer == num_layers - 1) ? Y : reserved_space_y;  // after activation/dropout
    mask = (layer == num_layers - 1 || p == 0) ? NULL : reserved_space_m;
    hidden = (layer == num_layers - 1) ? NULL : reserved_space_a; // before activation/dropout
    bias = BPtr[layer];
    int ifeat = (layer == 0) ? input_features : output_features[layer - 1];
    int ofeat = output_features[layer];

    float one = 1.f;
    float zero = 0.f;

    hipblasStatus_t cublas_status;
    // Call GEMM: fprop is Y = W'X
    cublas_status = mlp_gemm(
                        handle,
                        HIPBLAS_OP_T,
                        HIPBLAS_OP_N,
                        ofeat,
                        batch_size,
                        ifeat,
                        one,
                        weight,
                        ifeat,
                        input,
                        ifeat,
                        zero,
                        output,
                        ofeat);

    if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
    printf("GEMM fprop failed with %d\n", cublas_status);
    return 1;
    }

    const uint &input_size = ofeat;
    int num_blocks = 0;
    int num_SMs = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
    // Call biasReLU
    if (layer == (num_layers -1)) { // no activation
      hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, biasAdd_fprop<T>, BIAS_RELU_FW_NTHREADS, 0);
      biasAdd_fprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0, stream>>>(output, bias, batch_size, input_size);
    } else {  // GELU
      if (p == 0) {
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, biasAddGeLU_fprop<T>, BIAS_RELU_FW_NTHREADS, 0);
        biasAddGeLU_fprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0, stream>>>(output, hidden, bias,
                                                                                    batch_size, input_size);
      } else {
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, biasAddDropoutGeLU_fprop<T>, BIAS_RELU_FW_NTHREADS, 0);
        //number of times random will be generated per thread, to offset philox counter in thc random state
        int64_t counter_offset = ((input_size*batch_size-1)/(BIAS_RELU_FW_NTHREADS*num_SMs*num_blocks*ILP)+1)*ILP;
        std::pair<uint64_t, uint64_t> rng_engine_inputs;
        {
          std::lock_guard<std::mutex> lock(gen.mutex());
          rng_engine_inputs = at::check_generator<at::CUDAGeneratorImpl>(gen)->philox_engine_inputs(counter_offset);

        }
        biasAddDropoutGeLU_fprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0,
                             stream>>>(output, hidden, bias, mask, batch_size, input_size, p, rng_engine_inputs);
      }

    }

    // Set current output (after activation) as next layer input
    reserved_space_x = reserved_space_y;
    // Set next layer output

    if (layer < (num_layers -1)) {
        reserved_space_y += ofeat * batch_size;
        reserved_space_a += ofeat * batch_size;
        if (p > 0.0)
            reserved_space_m += ofeat * batch_size;
    }
  }

  return 0;
}

// Does a simple MLP bprop (GEMM+bias+ReLU).
// Needs reserved space to come back exactly as it was populated in fprop.
// Does dgrad and wgrad sequentially.
template <typename T>
int mlp_bp(
    T* X,
    T* Y,
    int input_features,
    int batch_size,
    T** WPtr,
    int num_layers,
    int* output_features,
    T* dY,
    T* reserved_space,
    T* reserved_activations,
    uint8_t* reserved_mask,
    T* work_space,
    T* dX,
    T** dwPtr,
    T** dbPtr,
    bool requires_grad,
    float p) {
  T* weight;
  T *dweight, *dx, *dy, *dbias;
  T *x, *y, *h;
  uint8_t *mask;
//  int activation = 1;

  // Where the dx of the biasReLU (== dy of gemm) is stored. Can be thrown away
  // after bp call.
  T* dy_gemm_base;
  // Where the dx after GEMM is stored.
  T* dx_gemm_base;
  // Where partial reduction results are stored.
  float* db_scratch;
  // Semaphores for reduction.
  int* semaphores;

  partition_mlp_bp_workspace<T>(
      batch_size,
      num_layers,
      output_features,
      work_space,
      &dy_gemm_base,
      &dx_gemm_base,
      &db_scratch,
      &semaphores);

  size_t semaphore_size = get_semaphores_size(num_layers, output_features) * sizeof(int);

  // Get cublas handle from Pytorch
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  // Get the stream from cublas handle to reuse for biasReLU kernel.
  hipStream_t stream;
  hipblasGetStream(handle, &stream);

  int* y_offsets = (int*)malloc(num_layers * sizeof(int));
  get_y_offsets(batch_size, num_layers, output_features, y_offsets);

  for (int layer = num_layers - 1; layer >= 0; layer--) {
    weight = WPtr[layer];
    dweight = dwPtr[layer];

    // x is read from reserved space
    x = (layer == 0) ? X : reserved_space + y_offsets[layer - 1];  // gemm + bias output

    // dx is written in workspace for all but layer==0
    dx = (layer == 0) ? dX : dx_gemm_base + y_offsets[layer - 1];

    // y is read from reserved space
    y = (layer == num_layers - 1) ? Y : reserved_space + y_offsets[layer];

    // note: last layer doesn't have h and mask
    h = (layer == num_layers - 1) ? NULL : reserved_activations + y_offsets[layer];  // activation + dropout output
    mask = ((layer == num_layers - 1) || (p == 0.0)) ? NULL : reserved_mask + y_offsets[layer];  // mask

    // dx from layer+1
    dy = (layer == num_layers - 1) ? dY : dx_gemm_base + y_offsets[layer];
    // dy_gemm is written to and read immediately
    T* dy_gemm = dy_gemm_base + y_offsets[layer];

    dbias = dbPtr[layer];
    int xfeat = (layer == 0) ? input_features : output_features[layer - 1];
    int yfeat = output_features[layer];

    float one = 1.f;
    float zero = 0.f;

    if (layer == (num_layers -1)) { // no activation
        // bgrad
        dim3 block(BIAS_RELU_BW_NTHREADS_X, BIAS_RELU_BW_NTHREADS_Y);
        int grid_x, grid_y;
        hipMemsetAsync(semaphores, 0, semaphore_size, stream);

        int block_x = BIAS_RELU_BW_NTHREADS_X;
        int block_y = BIAS_RELU_RED_PER_THREAD * BIAS_RELU_BW_NTHREADS_Y;
        get_biasAddRelu_bprop_grid_size(yfeat, batch_size, block_x, block_y, &grid_x, &grid_y);
        dim3 grid(grid_x, grid_y);
        biasAdd_bprop<T, 4><<<grid, block, 0, stream>>>(
          dy, yfeat, batch_size, db_scratch, semaphores, dbias);
        // bypass dgrad through reset pointer
        dy_gemm = dy;
    } else  { // gelu
        dim3 block(BIAS_RELU_BW_NTHREADS_X, BIAS_RELU_BW_NTHREADS_Y);
        int grid_x, grid_y;
        hipMemsetAsync(semaphores, 0, semaphore_size, stream);

        if (p == 0) {
            if(yfeat % (ILP * BIAS_RELU_BW_NTHREADS_X) == 0 &&
               is_aligned(y) &&
               is_aligned(h) &&
               is_aligned(dy) &&
               is_aligned(dy_gemm) &&
               is_aligned(dbias))
            {
              int block_x = ILP * BIAS_RELU_BW_NTHREADS_X;
              int block_y = BIAS_RELU_RED_PER_THREAD * BIAS_RELU_BW_NTHREADS_Y;
              // reusing the same grid size with biasAddRelu ... hopefully not a mistake
              get_biasAddRelu_bprop_grid_size(yfeat, batch_size, block_x, block_y, &grid_x, &grid_y);
              dim3 grid(grid_x, grid_y);
              biasAddGeLU_bprop_aligned<T, 4><<<grid, block, 0, stream>>>(
                y, h, dy, yfeat, batch_size, dy_gemm, db_scratch, semaphores, dbias);
            } else {
              int block_x = BIAS_RELU_BW_NTHREADS_X;
              int block_y = BIAS_RELU_RED_PER_THREAD * BIAS_RELU_BW_NTHREADS_Y;
              get_biasAddRelu_bprop_grid_size(yfeat, batch_size, block_x, block_y, &grid_x, &grid_y);
              dim3 grid(grid_x, grid_y);
              biasAddGeLU_bprop<T, 4><<<grid, block, 0, stream>>>(
                y, h, dy, yfeat, batch_size, dy_gemm, db_scratch, semaphores, dbias);
            }
        } else {
            if(yfeat % (ILP * BIAS_RELU_BW_NTHREADS_X) == 0 &&
               is_aligned(y) &&
               is_aligned(h) &&
               is_aligned(dy) &&
               is_aligned(dy_gemm) &&
               is_aligned(dbias))
            {
              int block_x = ILP * BIAS_RELU_BW_NTHREADS_X;
              int block_y = BIAS_RELU_RED_PER_THREAD * BIAS_RELU_BW_NTHREADS_Y;
              // reusing the same grid size with biasAddRelu ... hopefully not a mistake
              get_biasAddRelu_bprop_grid_size(yfeat, batch_size, block_x, block_y, &grid_x, &grid_y);
              dim3 grid(grid_x, grid_y);
              biasAddGeLUDropout_bprop_aligned<T, 4><<<grid, block, 0, stream>>>(
                y, h, dy, mask, yfeat, batch_size, dy_gemm, db_scratch, semaphores, dbias, p);
            } else {
              int block_x = BIAS_RELU_BW_NTHREADS_X;
              int block_y = BIAS_RELU_RED_PER_THREAD * BIAS_RELU_BW_NTHREADS_Y;
              get_biasAddRelu_bprop_grid_size(yfeat, batch_size, block_x, block_y, &grid_x, &grid_y);
              dim3 grid(grid_x, grid_y);
              biasAddGeLUDropout_bprop<T, 4><<<grid, block, 0, stream>>>(
                y, h, dy, mask, yfeat, batch_size, dy_gemm, db_scratch, semaphores, dbias, p);
            }
        }
    }
    hipblasStatus_t cublas_status;
    // Call GEMM dgrad
    if (layer > 0 || requires_grad == 1) {
      cublas_status = mlp_gemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        xfeat,
        batch_size,
        yfeat,
        one,
        weight,
        xfeat,
        dy_gemm,
        yfeat,
        zero,
        dx,
        xfeat);

      if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
        printf("GEMM dgrad failed with %d\n", cublas_status);
        return 1;
      }
    }

    // Call GEMM wgrad
    cublas_status = mlp_gemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        xfeat,
        yfeat,
        batch_size,
        one,
        x,
        xfeat,
        dy_gemm,
        yfeat,
        zero,
        dweight,
        xfeat);

    if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
      printf("GEMM wgrad failed with %d\n", cublas_status);
      return 1;
    }
  }

  return 0;
}



// Does a simple MLP bprop (GEMM+bias+ReLU).
// Needs reserved space to come back exactly as it was populated in fprop.
// Does dgrad and wgrad sequentially.
template <typename T>
int mlp_bp_input_only(
    T* X,
    T* Y,
    int input_features,
    int batch_size,
    T** WPtr,
    int num_layers,
    int* output_features,
    T* dY,
    T* reserved_space,
    T* reserved_activations,
    uint8_t* reserved_mask,
    T* work_space,
    T* dX,
    bool requires_grad,
    float p) {
  T* weight;
//  T *dweight, *dx, *dy, *dbias *x;
  T *dx, *dy;
  T *y, *h, *x;
  uint8_t *mask;

  // Where the dx of the biasReLU (== dy of gemm) is stored. Can be thrown away
  // after bp call.
  T* dy_gemm_base;
  // Where the dx after GEMM is stored.
  T* dx_gemm_base;
  // Where partial reduction results are stored.
  float* db_scratch;
  // Semaphores for reduction.
  int* semaphores;

  partition_mlp_bp_workspace<T>(
      batch_size,
      num_layers,
      output_features,
      work_space,
      &dy_gemm_base,
      &dx_gemm_base,
      &db_scratch,
      &semaphores);

  size_t semaphore_size = get_semaphores_size(num_layers, output_features) * sizeof(int);

  // Get cublas handle from Pytorch
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  // Get the stream from cublas handle to reuse for biasReLU kernel.
  hipStream_t stream;
  hipblasGetStream(handle, &stream);

  int* y_offsets = (int*)malloc(num_layers * sizeof(int));
  get_y_offsets(batch_size, num_layers, output_features, y_offsets);

  for (int layer = num_layers - 1; layer >= 0; layer--) {
    weight = WPtr[layer];
//    dweight = dwPtr[layer];

    // x is read from reserved space
    x = (layer == 0) ? X : reserved_space + y_offsets[layer - 1];  // gemm + bias output

    // dx is written in workspace for all but layer==0
    dx = (layer == 0) ? dX : dx_gemm_base + y_offsets[layer - 1];

    // y is read from reserved space
    y = (layer == num_layers - 1) ? Y : reserved_space + y_offsets[layer];

    // note: last layer doesn't have h and mask
    h = (layer == num_layers - 1) ? NULL : reserved_activations + y_offsets[layer];  // activation + dropout output
    mask = (layer == num_layers - 1) ? NULL : reserved_mask + y_offsets[layer];  // mask

    // dx from layer+1
    dy = (layer == num_layers - 1) ? dY : dx_gemm_base + y_offsets[layer];
    // dy_gemm is written to and read immediately
    T* dy_gemm = dy_gemm_base + y_offsets[layer];

//    dbias = dbPtr[layer];
    int xfeat = (layer == 0) ? input_features : output_features[layer - 1];
    int yfeat = output_features[layer];

    float one = 1.f;
    float zero = 0.f;

    if (layer == (num_layers -1)) { // no activation

        dy_gemm = dy;  // do nothing here because no need to backward to bias grad

    } else  { // gelu
//        dim3 block(BIAS_RELU_BW_NTHREADS_X, BIAS_RELU_BW_NTHREADS_Y);
//        int grid_x, grid_y;
//        hipMemsetAsync(semaphores, 0, semaphore_size, stream);
        int num_blocks = 0;
        int num_SMs = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

        if (p == 0) {
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, Gelu_bprop<T>, BIAS_RELU_FW_NTHREADS, 0);
            Gelu_bprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0, stream>>>(dy, h, y, yfeat, batch_size, dy_gemm);
//            if(yfeat % (ILP * BIAS_RELU_BW_NTHREADS_X) == 0 &&
//               is_aligned(y) &&
//               is_aligned(h) &&
//               is_aligned(dy) &&
//               is_aligned(dy_gemm) &&
//               is_aligned(dbias))
//            {
//              int block_x = ILP * BIAS_RELU_BW_NTHREADS_X;
//              int block_y = BIAS_RELU_RED_PER_THREAD * BIAS_RELU_BW_NTHREADS_Y;
//              // reusing the same grid size with biasAddRelu ... hopefully not a mistake
//              get_biasAddRelu_bprop_grid_size(yfeat, batch_size, block_x, block_y, &grid_x, &grid_y);
//              dim3 grid(grid_x, grid_y);
//              biasAddGeLU_bprop_aligned<T, 4><<<grid, block, 0, stream>>>(
//                y, h, dy, yfeat, batch_size, dy_gemm, db_scratch, semaphores, dbias);
//            } else {
//              int block_x = BIAS_RELU_BW_NTHREADS_X;
//              int block_y = BIAS_RELU_RED_PER_THREAD * BIAS_RELU_BW_NTHREADS_Y;
//              get_biasAddRelu_bprop_grid_size(yfeat, batch_size, block_x, block_y, &grid_x, &grid_y);
//              dim3 grid(grid_x, grid_y);
//              biasAddGeLU_bprop<T, 4><<<grid, block, 0, stream>>>(
//                y, h, dy, yfeat, batch_size, dy_gemm, db_scratch, semaphores, dbias);
//            }
        } else {
           hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, GeluDropout_bprop<T>, BIAS_RELU_FW_NTHREADS, 0);
           GeluDropout_bprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0, stream>>>(dy, h, y, mask, yfeat,
                                                                                       batch_size, dy_gemm, p);
//            if(yfeat % (ILP * BIAS_RELU_BW_NTHREADS_X) == 0 &&
//               is_aligned(y) &&
//               is_aligned(h) &&
//               is_aligned(dy) &&
//               is_aligned(dy_gemm) &&
//               is_aligned(dbias))
//            {
//              int block_x = ILP * BIAS_RELU_BW_NTHREADS_X;
//              int block_y = BIAS_RELU_RED_PER_THREAD * BIAS_RELU_BW_NTHREADS_Y;
//              // reusing the same grid size with biasAddRelu ... hopefully not a mistake
//              get_biasAddRelu_bprop_grid_size(yfeat, batch_size, block_x, block_y, &grid_x, &grid_y);
//              dim3 grid(grid_x, grid_y);
//              biasAddGeLUDropout_bprop_aligned<T, 4><<<grid, block, 0, stream>>>(
//                y, h, dy, mask, yfeat, batch_size, dy_gemm, db_scratch, semaphores, dbias, p);
//            } else {
//              int block_x = BIAS_RELU_BW_NTHREADS_X;
//              int block_y = BIAS_RELU_RED_PER_THREAD * BIAS_RELU_BW_NTHREADS_Y;
//              get_biasAddRelu_bprop_grid_size(yfeat, batch_size, block_x, block_y, &grid_x, &grid_y);
//              dim3 grid(grid_x, grid_y);
//              biasAddGeLUDropout_bprop<T, 4><<<grid, block, 0, stream>>>(
//                y, h, dy, mask, yfeat, batch_size, dy_gemm, db_scratch, semaphores, dbias, p);
//            }
        }
    }
    hipblasStatus_t cublas_status;
    // Call GEMM dgrad only
    if (layer > 0 || requires_grad == 1) {
      cublas_status = mlp_gemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        xfeat,
        batch_size,
        yfeat,
        one,
        weight,
        xfeat,
        dy_gemm,
        yfeat,
        zero,
        dx,
        xfeat);

      if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
        printf("GEMM dgrad failed with %d\n", cublas_status);
        return 1;
      }
    }

    if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
      printf("GEMM wgrad failed with %d\n", cublas_status);
      return 1;
    }
  }

  return 0;
}

// Instantiate for floating point types
template int mlp_fp<float>(
    float* X,
    int input_features,
    int batch_size,
    float** WPtr,
    int num_layers,
    int* output_features,
    float** BPtr,
    float* Y,
    float* reserved_space,
    float* reserved_activations,
    uint8_t* reserved_mask,
    float p);

template int mlp_bp<float>(
    float* X,
    float* Y,
    int input_features,
    int batch_size,
    float** WPtr,
    int num_layers,
    int* output_features,
    float* dY,
    float* reserved_space,
    float* reserved_activations,
    uint8_t* reserved_mask,
    float* work_space,
    float* dX,
    float** dwPtr,
    float** dbPtr,
    bool requires_grad,
    float p);

template int mlp_bp_input_only<float>(
    float* X,
    float* Y,
    int input_features,
    int batch_size,
    float** WPtr,
    int num_layers,
    int* output_features,
    float* dY,
    float* reserved_space,
    float* reserved_activations,
    uint8_t* reserved_mask,
    float* work_space,
    float* dX,
    bool requires_grad,
    float p);

template int mlp_fp<at::Half>(
    at::Half* X,
    int input_features,
    int batch_size,
    at::Half** WPtr,
    int num_layers,
    int* output_features,
    at::Half** BPtr,
    at::Half* Y,
    at::Half* reserved_space,
    at::Half* reserved_activations,
    uint8_t* reserved_mask,
    float p);

template int mlp_bp<at::Half>(
    at::Half* X,
    at::Half* Y,
    int input_features,
    int batch_size,
    at::Half** WPtr,
    int num_layers,
    int* output_features,
    at::Half* dY,
    at::Half* reserved_space,
    at::Half* reserved_activations,
    uint8_t* reserved_mask,
    at::Half* work_space,
    at::Half* dX,
    at::Half** dwPtr,
    at::Half** dbPtr,
    bool requires_grad,
    float p);

template int mlp_bp_input_only<at::Half>(
    at::Half* X,
    at::Half* Y,
    int input_features,
    int batch_size,
    at::Half** WPtr,
    int num_layers,
    int* output_features,
    at::Half* dY,
    at::Half* reserved_space,
    at::Half* reserved_activations,
    uint8_t* reserved_mask,
    at::Half* work_space,
    at::Half* dX,
    bool requires_grad,
    float p);

template int mlp_fp<double>(
    double* X,
    int input_features,
    int batch_size,
    double** WPtr,
    int num_layers,
    int* output_features,
    double** BPtr,
    double* Y,
    double* reserved_space,
    double* reserved_activations,
    uint8_t* reserved_mask,
    float p);

template int mlp_bp<double>(
    double* X,
    double* Y,
    int input_features,
    int batch_size,
    double** WPtr,
    int num_layers,
    int* output_features,
    double* dY,
    double* reserved_space,
    double* reserved_activations,
    uint8_t* reserved_mask,
    double* work_space,
    double* dX,
    double** dwPtr,
    double** dbPtr,
    bool requires_grad,
    float p);

template int mlp_bp_input_only<double>(
    double* X,
    double* Y,
    int input_features,
    int batch_size,
    double** WPtr,
    int num_layers,
    int* output_features,
    double* dY,
    double* reserved_space,
    double* reserved_activations,
    uint8_t* reserved_mask,
    double* work_space,
    double* dX,
    bool requires_grad,
    float p);



template size_t get_mlp_bp_workspace_in_bytes<float>(
    int batch_size,
    int num_layers,
    const int* output_features);
template size_t get_mlp_bp_workspace_in_bytes<at::Half>(
    int batch_size,
    int num_layers,
    const int* output_features);
template size_t get_mlp_bp_workspace_in_bytes<double>(
    int batch_size,
    int num_layers,
    const int* output_features);
