#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <torch/torch.h>
#include <cmath>
#include <math.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#ifdef OLD_GENERATOR_PATH
#include <ATen/CUDAGeneratorImpl.h>
#else
#include <ATen/cuda/CUDAGeneratorImpl.h>
#endif
#include <hiprand/hiprand_kernel.h>

// includes cublaslt
#include <hipblaslt.h>
#include "cublaslt_wrapper.cuh"

// constants for fused bias+relu kernel
#define BIAS_RELU_FW_NTHREADS 256 // forward number of thread per block
#define BIAS_RELU_BW_NTHREADS_X 32 // backward number of thread in feature dim
#define BIAS_RELU_BW_NTHREADS_Y 16 // backward number of thread in batch dim
#define BIAS_RELU_RED_PER_THREAD 16 // backward minimal reduction length per thread

// move to a header later on
#define ILP 4
#define BACKCOEFF M_2_SQRTPI * M_SQRT1_2 * 0.5f
template<typename T>
__host__ __device__ __forceinline__ bool is_aligned(T* p){
  return ((uint64_t)p) % (ILP*sizeof(T)) == 0;
}

template<typename T>
__device__ __forceinline__ void load_store(T* dst, T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}
template<typename T>
__device__ __forceinline__ void load_store(T* dst, volatile T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}
template<typename T>
__device__ __forceinline__ void load_store(volatile T* dst, T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}

// Keep ReLU in float only. When using half, cast to float before calling.
__device__ __inline__ float relu(float a) {
  float retf = max(a, 0.f);
  return (retf);
}


// Keep gelu in float only. When using half, cast to float before calling.
__device__ __inline__ float gelu(float a) {
  float retf = a * normcdff(a);
  return (retf);
}


// Keep gelu in float only. When using half, cast to float before calling.
__device__ __inline__ float gelu_back(float dy, float a) {

  // dy is the gradient w.r.t the gelu output
  float cdf = normcdff(a);
  float pdf = BACKCOEFF * expf(-0.5f * a * a);
  float retf = cdf + a * pdf;

  return (dy * retf);
}




// FP64 Wrapper around cublas GEMMEx
hipblasStatus_t mlp_gemm(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    float alpha,
    const double* A,
    int lda,
    const double* B,
    int ldb,
    const float beta,
    double* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      &alpha,
      A,
      HIP_R_64F,
      lda,
      B,
      HIP_R_64F,
      ldb,
      &beta,
      C,
      HIP_R_64F,
      ldc,
      HIP_R_64F,
      HIPBLAS_GEMM_DEFAULT);
}

// FP32 Wrapper around cublas GEMMEx
hipblasStatus_t mlp_gemm(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    float alpha,
    const float* A,
    int lda,
    const float* B,
    int ldb,
    const float beta,
    float* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      &alpha,
      A,
      HIP_R_32F,
      lda,
      B,
      HIP_R_32F,
      ldb,
      &beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      HIPBLAS_GEMM_DEFAULT);
}

// FP16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t mlp_gemm(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    float alpha,
    const at::Half* A,
    int lda,
    const at::Half* B,
    int ldb,
    float beta,
    at::Half* C,
    int ldc) {

  const half halpha = __float2half_rn(alpha);
  const half hbeta = __float2half_rn(beta);
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      &halpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      &hbeta,
      C,
      HIP_R_16F,
      ldc,
      HIPBLAS_COMPUTE_16F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

///////// CUBLAS LT FUNCTIONS /////////////


/////////////////////////////////////////////////////////////////////////////////////////////////////

// Bias ADD + ReLU. Assume input X is [features x batch size], column major.
// Activation support fuesed ReLU. Safe to call in-place.
template <typename T>
__global__ void DropoutGeLU_fprop(T *X, T *Y, uint8_t *mask, uint batch_size, uint features, float p,
                                         std::pair<uint64_t, uint64_t> seeds) {
  T r_x[ILP];
  T r_y[ILP];
  uint8_t r_m[ILP];
//  uint8_t r_m[ILP];
  float pinv = 1.f/(1.f-p);

  if(is_aligned(X) && is_aligned(Y) && features % ILP ==0) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(
        seeds.first,
        tid,
        seeds.second,
        &state);

    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      load_store(r_x, X, 0 , tid);
      load_store(r_y, Y, 0 , tid);
      load_store(r_m, mask, 0, tid);  // mask has the same size with X

      float4 rand = hiprand_uniform4(&state);
      rand.x = rand.x >= p;
      rand.y = rand.y >= p;
      rand.z = rand.z >= p;
      rand.w = rand.w >= p;

#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]);
        r_y[ii] = bias_sum;  // store the mm + bias output
        r_x[ii] = gelu(bias_sum) * (float)(&rand.x)[ii]*pinv;  // gelu * dropout mask
        r_m[ii] = (uint8_t)(&rand.x)[ii];  // store the mask values in buffer
      }
      load_store(X, r_x, tid , 0);
      load_store(mask, r_m, tid , 0);
      load_store(Y, r_y, tid , 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(
        seeds.first,
        tid,
        seeds.second,
        &state);

    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {

      float4 rand = hiprand_uniform4(&state);
      rand.x = rand.x >= p;
      rand.y = rand.y >= p;
      rand.z = rand.z >= p;
      rand.w = rand.w >= p;
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          r_x[ii] = X[idx];
          r_m[ii] = mask[idx];
          r_y[ii] = Y[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]);
        r_y[ii] = bias_sum;
        r_x[ii] = gelu(bias_sum)*(float)(&rand.x)[ii]*pinv;
        r_m[ii] = (uint8_t)(&rand.x)[ii];
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          X[idx] = r_x[ii];
          mask[idx] = r_m[ii];
          Y[idx] = r_y[ii];
        }
      }
    }
  }
}

template <typename T>
__global__ void GeLU_fprop(T *X, T *Y, uint batch_size, uint features) {
  T r_x[ILP];
  T r_y[ILP];

  if(is_aligned(X) && is_aligned(Y) && features % ILP ==0) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      load_store(r_x, X, 0 , tid);
      load_store(r_y, Y, 0 , tid);

#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]);
        //r_x[ii] = relu(bias_sum)*(&rand.x)[ii]*pinv;
        r_y[ii] = bias_sum;  // store the mm + bias output
        r_x[ii] = gelu(bias_sum);  // gelu * dropout mask
      }
      load_store(X, r_x, tid , 0);
      load_store(Y, r_y, tid , 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {

#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          r_x[ii] = X[idx];
          r_y[ii] = Y[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]);
        r_y[ii] = bias_sum;
        r_x[ii] = gelu(bias_sum);
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          X[idx] = r_x[ii];
          Y[idx] = r_y[ii];
        }
      }
    }
  }
}


// Bias ADD + ReLU. Assume input X is [features x batch size], column major.
// Activation support fuesed ReLU. Safe to call in-place.
template <typename T>
__global__ void DropoutGeLU_presampled_fprop(T *X, T *Y, uint8_t *mask, uint batch_size, uint features, float p) {
  T r_x[ILP];
  T r_y[ILP];
  uint8_t r_m[ILP];
//  uint8_t r_m[ILP];
  float pinv = 1.f/(1.f-p);

  if(is_aligned(X) && is_aligned(Y) && features % ILP ==0) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      load_store(r_x, X, 0 , tid);
      load_store(r_y, Y, 0 , tid);
      load_store(r_m, mask, 0, tid);  // mask has the same size with X

#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]);
        r_y[ii] = bias_sum;  // store the mm + bias output
        r_x[ii] = gelu(bias_sum) * (float)(r_m[ii]) *pinv;  // gelu * dropout mask
      }
      load_store(X, r_x, tid , 0);  // X stores gelu output
      load_store(Y, r_y, tid , 0);  // Y stores gelu input
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          r_x[ii] = X[idx];
          r_m[ii] = mask[idx];
          r_y[ii] = Y[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float bias_sum = static_cast<float>(r_x[ii]);
        r_y[ii] = bias_sum;
        r_x[ii] = gelu(bias_sum)*(float)(r_m[ii])*pinv;
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          X[idx] = r_x[ii];
          Y[idx] = r_y[ii];
        }
      }
    }
  }
}


//////////////////////// BACKPROP///////////////////////////////////////


// ReLU. Assume input X is [features x batch size], column major.
// Safe to call in-place.
template <typename T>
__global__ void Gelu_bprop(T *dY, T* H, T *Y, uint features, uint batch_size, T *dX) {
  T r_dy[ILP];
//  T r_y[ILP];
  T r_h[ILP];
  if(is_aligned(dY) &&
     is_aligned(Y) &&
     is_aligned(H) &&
     is_aligned(dX) &&
     features % ILP ==0) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      load_store(r_dy, dY, 0 , tid);
//      load_store(r_y, Y, 0 , tid);
      load_store(r_h, H, 0 , tid);
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
          r_dy[ii] = gelu_back((float)r_dy[ii], (float)r_h[ii]);
      }
      load_store(dX, r_dy, tid, 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          r_dy[ii] = dY[idx];
//          r_y[ii] = Y[idx];
          r_h[ii] = H[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        r_dy[ii] = gelu_back((float)r_dy[ii], (float)r_h[ii]);
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          dX[idx] = r_dy[ii];
        }
      }
    }
  }
}


// ReLU. Assume input X is [features x batch size], column major.
// Safe to call in-place.
template <typename T>
__global__ void GeluDropout_bprop(T *dY, T* H, T *Y, uint8_t* mask, uint features, uint batch_size, T *dX, float p) {
  T r_dy[ILP];
//  T r_y[ILP];
  T r_h[ILP];
  uint8_t r_m[ILP];
  float pinv = 1.0f / (1.0f - p);
  if(is_aligned(dY) &&
     is_aligned(Y) &&
     is_aligned(H) &&
     is_aligned(dX) &&
     features % ILP ==0) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      load_store(r_dy, dY, 0 , tid);
//      load_store(r_y, Y, 0 , tid);
      load_store(r_h, H, 0 , tid);
      load_store(r_m, mask, 0 , tid);
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
            r_dy[ii] = gelu_back((float)r_dy[ii] * (float)r_m[ii] * pinv, (float)r_h[ii]);
      }
      load_store(dX, r_dy, tid, 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          r_dy[ii] = dY[idx];
//          r_y[ii] = Y[idx];
          r_h[ii] = H[idx];
          r_m[ii] = mask[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        r_dy[ii] = gelu_back( (float)r_dy[ii] * (float)r_m[ii] * pinv, (float)r_h[ii] );
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          dX[idx] = r_dy[ii];
        }
      }
    }
  }
}




// Lists where the num_layers-1 intermediate Y buffers start in reserved space on fprop, starting
// offset 0. The last Y value is, of course, stored in the user provided output buffer.
void get_y_offsets(
    int batch_size,
    int num_layers,
    const int* output_features,
    int* y_start_offsets) {
  y_start_offsets[0] = 0;
  for (int i = 1; i < num_layers; i++) {
    y_start_offsets[i] = y_start_offsets[i - 1] + batch_size * output_features[i - 1];
  }
}

// Returns the size of all fprop activations combined
size_t get_all_activations_size(int64_t batch_size, int num_layers, const int* output_features) {
  size_t acts_size = 0;
  for (int l = 0; l < num_layers; l++) {
    acts_size += output_features[l] * batch_size;
  }
  return acts_size;
}

// Returns the reserved space (in elements) needed for the MLP
size_t get_mlp_reserved_space(int64_t batch_size, int num_layers, const int* output_features) {
  size_t res_space = 0;
  // Need to store output of every intermediate MLP - size equal to output_features[i] * batch_size
  // for all 'i' in [0, num_layers-1)
  for (int l = 0; l < num_layers - 1; l++) {
    res_space += output_features[l] * batch_size;
  }
  return res_space;
}

// Returns the size of all fprop activations combined
// no dropout and activation at the last layer so no need that one
size_t get_mlp_activation_space(int64_t batch_size, int num_layers, const int* output_features) {
  size_t acts_size = 0;
  for (int l = 0; l < num_layers - 1; l++) {
    acts_size += output_features[l] * batch_size;
  }
  return acts_size;
}

#if 0
// Returns the work space (in elements) needed for the MLP bprop.
size_t get_mlp_bp_workspace (int batch_size, int num_layers, const int* output_features) {
    /*
       Workspace is partitioned as
       DY_GEMMs : DX_GEMMs
    */
    size_t work_space = 0;

    // Store each intermediate dY explicitly. Need 2 dYs per MLP layer (one for o/p
    // of biasReLU_bp and one for o/p of dgrad GEMM).
    work_space += 2*get_all_activations_size(batch_size, num_layers, output_features);

    return work_space;
}
#endif

// Returns the work space (in elements) needed for the MLP bprop.
template <typename T>
size_t get_mlp_bp_workspace_in_bytes(int batch_size, int num_layers, const int* output_features) {
  size_t work_space = 0;

  // Store each intermediate dY explicitly. Need 2 dYs per MLP layer (one for o/p
  // of biasReLU_bp and one for o/p of dgrad GEMM).
  work_space += 2 * get_all_activations_size(batch_size, num_layers, output_features) * sizeof(T);

  return work_space;
}

// Returns pointers to each segment of the workspace
template <typename T>
void partition_mlp_bp_workspace(
    int batch_size,
    int num_layers,
    const int* output_features,
    void* work_space,
    T** dy_gemms,
    T** dx_gemms) {
  /*
     Workspace is partitioned as
     DY_GEMMs : DX_GEMMs : DB_SCRATCH : SEMAPHORES
  */
  // Start address where dy_gemm tensors are stored
  *dy_gemms = reinterpret_cast<T*>(work_space);
  // Start address where dx_gemm tensors are stored
  *dx_gemms = *dy_gemms + get_all_activations_size(batch_size, num_layers, output_features);
  // Start address where db intermediate tensors are stored

  return;
}

// Does a simple MLP fprop (GEMM+bias+ReLU).
// Can handle num_layers number of layers, each with its own shape. Output of layer i is assumed
// to be input of layer i+1. output_features, WPtr and BPtr are arrays of length num_layers, and
// must be in the same order i.e. WPtr[i] and BPtr[i] are respectively the weight and bias of layer
// 'i'.
template <typename T>
int mlp_fp(
    T* X,
    int input_features,
    int batch_size,
    T** WPtr,
    T** BPtr,
    int num_layers,
    int* output_features,
    T* Y,
    T* reserved_space,
    T* reserved_activations,
    uint8_t* reserved_mask,
    void* lt_workspace,
    float p) {
  auto gen = at::cuda::detail::getDefaultCUDAGenerator();
  T *weight, *input, *output, *hidden, *bias;
  uint8_t *mask,  *reserved_space_m;
  T *reserved_space_x, *reserved_space_y, *reserved_space_a;
  reserved_space_x = NULL;
  reserved_space_a = reserved_activations;
  reserved_space_y = reserved_space;
  reserved_space_m = reserved_mask;

  // Get cublas handle from Pytorch
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  // Get the strea* from cublas handle to reuse for biasReLU kernel.
  hipStream_t stream;
  hipblasGetStream(handle, &stream);

  for (int layer = 0; layer < num_layers; layer++) {
    weight = WPtr[layer];
    input = (layer == 0) ? X : reserved_space_x;
    output = (layer == num_layers - 1) ? Y : reserved_space_y;  // after activation/dropout
    mask = (layer == num_layers - 1 || p == 0) ? NULL : reserved_space_m;
    hidden = (layer == num_layers - 1) ? NULL : reserved_space_a; // before activation/dropout
    bias = BPtr[layer];
    int ifeat = (layer == 0) ? input_features : output_features[layer - 1];
    int ofeat = output_features[layer];

    float one = 1.f;
    float zero = 0.f;

    int cublas_status;
    // Call GEMM: fprop is Y = W'X

    // gelu is only applied in the middle layers and without dropout
    bool use_gelu = (layer < (num_layers - 1) && p == 0);

    if (use_gelu)
        cublas_status = gemm_bias_gelu_lt(
            (hipblasLtHandle_t)handle,
            HIPBLAS_OP_T,
            HIPBLAS_OP_N,
            ofeat,
            batch_size,
            ifeat,
            &one, /* host pointer */
            weight,
            ifeat,
            input,
            ifeat,
            &zero, /* host pointer */
            output,
            ofeat,
            lt_workspace,
            1 << 22,
            stream,
            true,
            static_cast<const void*>(hidden),
            static_cast<const void*>(bias));

    else
        cublas_status = gemm_bias_lt(
            (hipblasLtHandle_t)handle,
            HIPBLAS_OP_T,
            HIPBLAS_OP_N,
            ofeat,
            batch_size,
            ifeat,
            &one, /* host pointer */
            weight,
            ifeat,
            input,
            ifeat,
            &zero, /* host pointer */
            output,
            ofeat,
            lt_workspace,
            1 << 22,
            stream,
            true,
            static_cast<const void*>(bias));

    if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
        printf("GEMM fprop failed with %d\n", cublas_status);
        return 1;
    }

    const uint &input_size = ofeat;
    int num_blocks = 0;
    int num_SMs = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
    // Call biasReLU
    if (layer == (num_layers -1)) { // no activation
      // do nothing here
    } else {  // GELU
      if (p == 0) {
           // gelu has already been applied at mlp_gemm
//         hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, GeLU_fprop<T>, BIAS_RELU_FW_NTHREADS, 0);
//         GeLU_fprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0, stream>>>(output, hidden,
//                                                                                     batch_size, input_size);
      } else {
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, DropoutGeLU_fprop<T>, BIAS_RELU_FW_NTHREADS, 0);
        //number of times random will be generated per thread, to offset philox counter in thc random state
        int64_t counter_offset = ((input_size*batch_size-1)/(BIAS_RELU_FW_NTHREADS*num_SMs*num_blocks*ILP)+1)*ILP;
        std::pair<uint64_t, uint64_t> rng_engine_inputs;
        {
          std::lock_guard<std::mutex> lock(gen.mutex());
          rng_engine_inputs = at::check_generator<at::CUDAGeneratorImpl>(gen)->philox_engine_inputs(counter_offset);

        }
        // actually its gelu -> dropout
        DropoutGeLU_fprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0,
                             stream>>>(output, hidden, mask, batch_size, input_size, p, rng_engine_inputs);
      }

    }

    // Set current output (after activation) as next layer input
    reserved_space_x = reserved_space_y;
    // Set next layer output

    if (layer < (num_layers -1)) {
        reserved_space_y += ofeat * batch_size;
        reserved_space_a += ofeat * batch_size;
        if (p > 0.0)
            reserved_space_m += ofeat * batch_size;
    }
  }

  return 0;
}

// Does a simple MLP bprop (GEMM+bias+ReLU).
// Needs reserved space to come back exactly as it was populated in fprop.
// Does dgrad and wgrad sequentially.
template <typename T>
int mlp_bp(
    T* X,
    T* Y,
    int input_features,
    int batch_size,
    T** WPtr,
    T** BPtr,
    int num_layers,
    int* output_features,
    T* dY,
    T* reserved_space,
    T* reserved_activations,
    uint8_t* reserved_mask,
    T* work_space,
    T* dX,
    T** dwPtr,
    T** dbPtr,
    bool requires_grad,
    void* lt_workspace,
    float p,
    bool recompute) {
  T *weight, *input, *output, *hidden, *bias;
  T *dweight, *dx, *dy, *dbias;
  T *x, *y, *h;
  uint8_t *mask;
  float one = 1.f;
  float zero = 0.f;
  int cublas_status;
  int xfeat, yfeat, ofeat, ifeat;

  // if recompute: reserved space, reserved activation have to be recompute from X, Weights and biases

  // Get cublas handle from Pytorch
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  // Get the stream from cublas handle to reuse for biasReLU kernel.
  hipStream_t stream;
  hipblasGetStream(handle, &stream);

  if (recompute)    {
    uint8_t *mask,  *reserved_space_m;
    T *reserved_space_x, *reserved_space_y, *reserved_space_a;
    reserved_space_x = NULL;
    reserved_space_a = reserved_activations;
    reserved_space_y = reserved_space;
    reserved_space_m = reserved_mask;

    for (int layer = 0; layer < num_layers; layer++) {
        weight = WPtr[layer];
        input = (layer == 0) ? X : reserved_space_x;
        output = (layer == num_layers - 1) ? Y : reserved_space_y;  // after activation/dropout
        mask = (layer == num_layers - 1 || p == 0) ? NULL : reserved_space_m;
        hidden = (layer == num_layers - 1) ? NULL : reserved_space_a; // before activation/dropout
        bias = BPtr[layer];
        ifeat = (layer == 0) ? input_features : output_features[layer - 1];
        ofeat = output_features[layer];

        bool use_gelu = (layer < (num_layers - 1) && p == 0);

        if (use_gelu)
            cublas_status = gemm_bias_gelu_lt(
                (hipblasLtHandle_t)handle,
                HIPBLAS_OP_T,
                HIPBLAS_OP_N,
                ofeat,
                batch_size,
                ifeat,
                &one, /* host pointer */
                weight,
                ifeat,
                input,
                ifeat,
                &zero, /* host pointer */
                output,
                ofeat,
                lt_workspace,
                1 << 22,
                stream,
                true,
                static_cast<const void*>(hidden),
                static_cast<const void*>(bias));

        else
            cublas_status = gemm_bias_lt(
                (hipblasLtHandle_t)handle,
                HIPBLAS_OP_T,
                HIPBLAS_OP_N,
                ofeat,
                batch_size,
                ifeat,
                &one, /* host pointer */
                weight,
                ifeat,
                input,
                ifeat,
                &zero, /* host pointer */
                output,
                ofeat,
                lt_workspace,
                1 << 22,
                stream,
                true,
                static_cast<const void*>(bias));

        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            printf("GEMM fprop failed with %d\n", cublas_status);
            return 1;
        }

        const uint &input_size = ofeat;
        int num_blocks = 0;
        int num_SMs = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

        if (layer == (num_layers -1)) { // no activation
          // do nothing here
        } else {  // GELU
          if (p == 0) {
               // gelu has already been applied at mlp_gemm
          } else {
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, DropoutGeLU_presampled_fprop<T>,
                                                          BIAS_RELU_FW_NTHREADS, 0);
            //number of times random will be generated per thread, to offset philox counter in thc random state
            int64_t counter_offset = ((input_size*batch_size-1)/(BIAS_RELU_FW_NTHREADS*num_SMs*num_blocks*ILP)+1)*ILP;
            // actually its gelu -> dropout
            DropoutGeLU_presampled_fprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0,
                                 stream>>>(output, hidden, mask, batch_size, input_size, p);
          }
        }

        // Set current output (after activation) as next layer input
        reserved_space_x = reserved_space_y;
        // Set next layer output

        if (layer < (num_layers -1)) {
            reserved_space_y += ofeat * batch_size;
            reserved_space_a += ofeat * batch_size;
            if (p > 0.0)
                reserved_space_m += ofeat * batch_size;
        }

    }

  }

  // Backward starts

  // Where the dx of the biasReLU (== dy of gemm) is stored. Can be thrown away
  // after bp call.
  T* dy_gemm_base;
  // Where the dx after GEMM is stored.
  T* dx_gemm_base;

  partition_mlp_bp_workspace<T>(
      batch_size,
      num_layers,
      output_features,
      work_space,
      &dy_gemm_base,
      &dx_gemm_base);

  int* y_offsets = (int*)malloc(num_layers * sizeof(int));
  get_y_offsets(batch_size, num_layers, output_features, y_offsets);

  for (int layer = num_layers - 1; layer >= 0; layer--) {
    weight = WPtr[layer];
    dweight = dwPtr[layer];

    // x is read from reserved space
    x = (layer == 0) ? X : reserved_space + y_offsets[layer - 1];  // gemm + bias output

    // dx is written in workspace for all but layer==0
    dx = (layer == 0) ? dX : dx_gemm_base + y_offsets[layer - 1];

    // y is read from reserved space
    y = (layer == num_layers - 1) ? Y : reserved_space + y_offsets[layer];

    // note: last layer doesn't have h and mask
    h = (layer == num_layers - 1) ? NULL : reserved_activations + y_offsets[layer];  // activation + dropout output
    mask = ((layer == num_layers - 1) || (p == 0.0)) ? NULL : reserved_mask + y_offsets[layer];  // mask

    // dx from layer+1
    dy = (layer == num_layers - 1) ? dY : dx_gemm_base + y_offsets[layer];
    // dy_gemm is written to and read immediately
    T* dy_gemm = dy_gemm_base + y_offsets[layer];

    dbias = dbPtr[layer];
    xfeat = (layer == 0) ? input_features : output_features[layer - 1];
    yfeat = output_features[layer];

    if (layer == (num_layers - 1)) { // no activation
        // Don't need to do anything
        // bypass dgrad through reset pointer
        dy_gemm = dy;
    } else  { // gelu
        int num_blocks = 0;
        int num_SMs = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

        if (p == 0) {
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, Gelu_bprop<T>, BIAS_RELU_FW_NTHREADS, 0);
            Gelu_bprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0, stream>>>(dy, h, y, yfeat, batch_size, dy_gemm);
        } else {
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, GeluDropout_bprop<T>, BIAS_RELU_FW_NTHREADS, 0);
            GeluDropout_bprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0, stream>>>(dy, h, y, mask, yfeat,
                                                                                       batch_size, dy_gemm, p);
        }
    }
    hipblasStatus_t cublas_status;
    // Call GEMM dgrad
    if (layer > 0 || requires_grad == 1) {
      cublas_status = mlp_gemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        xfeat,
        batch_size,
        yfeat,
        one,
        weight,
        xfeat,
        dy_gemm,
        yfeat,
        zero,
        dx,
        xfeat);

      if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
        printf("GEMM dgrad failed with %d\n", cublas_status);
        return 1;
      }
    }

    // Call GEMM wgrad and bgrad
    int cublaslt_status_ = 1;
    cublaslt_status_ = gemm_bgradb_lt(
        (hipblasLtHandle_t)handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        xfeat,
        yfeat,
        batch_size,
        &one, /* host pointer */
        x,
        xfeat,
        dy_gemm,
        yfeat,
        &zero, /* host pointer */
        dweight,
        xfeat,
        lt_workspace,
        1 << 22,
        stream,
        true,
        static_cast<void*>(dbias));

    if (cublaslt_status_ != HIPBLAS_STATUS_SUCCESS) {
      printf("GEMM wgrad failed with %d\n", cublas_status);
      return 1;
    }
  }

  return 0;
}


template <typename T>
int mlp_bp_input_only(
    T* X,
    T* Y,
    int input_features,
    int batch_size,
    T** WPtr,
    int num_layers,
    int* output_features,
    T* dY,
    T* reserved_space,
    T* reserved_activations,
    uint8_t* reserved_mask,
    T* work_space,
    T* dX,
    bool requires_grad,
    void* lt_workspace,
    float p) {
  T* weight;
  T *dx, *dy;
  T *x, *y, *h;
  uint8_t *mask;
//  int activation = 1;

  // Where the dx of the biasReLU (== dy of gemm) is stored. Can be thrown away
  // after bp call.
  T* dy_gemm_base;
  // Where the dx after GEMM is stored.
  T* dx_gemm_base;

  partition_mlp_bp_workspace<T>(
      batch_size,
      num_layers,
      output_features,
      work_space,
      &dy_gemm_base,
      &dx_gemm_base);

  // Get cublas handle from Pytorch
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  // Get the stream from cublas handle to reuse for biasReLU kernel.
  hipStream_t stream;
  hipblasGetStream(handle, &stream);

  int* y_offsets = (int*)malloc(num_layers * sizeof(int));
  get_y_offsets(batch_size, num_layers, output_features, y_offsets);

  for (int layer = num_layers - 1; layer >= 0; layer--) {
    weight = WPtr[layer];
//     dweight = dwPtr[layer];

    // x is read from reserved space
    x = (layer == 0) ? X : reserved_space + y_offsets[layer - 1];  // gemm + bias output

    // dx is written in workspace for all but layer==0
    dx = (layer == 0) ? dX : dx_gemm_base + y_offsets[layer - 1];

    // y is read from reserved space
    y = (layer == num_layers - 1) ? Y : reserved_space + y_offsets[layer];

    // note: last layer doesn't have h and mask
    h = (layer == num_layers - 1) ? NULL : reserved_activations + y_offsets[layer];  // activation + dropout output
    mask = ((layer == num_layers - 1) || (p == 0.0)) ? NULL : reserved_mask + y_offsets[layer];  // mask

    // dx from layer+1
    dy = (layer == num_layers - 1) ? dY : dx_gemm_base + y_offsets[layer];
    // dy_gemm is written to and read immediately
    T* dy_gemm = dy_gemm_base + y_offsets[layer];

//     dbias = dbPtr[layer];
    int xfeat = (layer == 0) ? input_features : output_features[layer - 1];
    int yfeat = output_features[layer];

    float one = 1.f;
    float zero = 0.f;

    if (layer == (num_layers -1)) { // no activation

        dy_gemm = dy;
    } else  { // gelu
        int num_blocks = 0;
        int num_SMs = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

        if (p == 0) {
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, Gelu_bprop<T>, BIAS_RELU_FW_NTHREADS, 0);
            Gelu_bprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0, stream>>>(dy, h, y, yfeat, batch_size, dy_gemm);
        } else {
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, GeluDropout_bprop<T>, BIAS_RELU_FW_NTHREADS, 0);
            GeluDropout_bprop<<<num_SMs*num_blocks, BIAS_RELU_FW_NTHREADS, 0, stream>>>(dy, h, y, mask, yfeat,
                                                                                       batch_size, dy_gemm, p);
        }
    }
    hipblasStatus_t cublas_status;
    // Call GEMM dgrad
    if (layer > 0 || requires_grad == 1) {
      cublas_status = mlp_gemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        xfeat,
        batch_size,
        yfeat,
        one,
        weight,
        xfeat,
        dy_gemm,
        yfeat,
        zero,
        dx,
        xfeat);

      if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
        printf("GEMM dgrad failed with %d\n", cublas_status);
        return 1;
      }
    }

    // No need to Call GEMM wgrad and bgrad

  }

  return 0;
}






// Instantiate for floating point types
template int mlp_fp<float>(
    float* X,
    int input_features,
    int batch_size,
    float** WPtr,
    float** BPtr,
    int num_layers,
    int* output_features,
    float* Y,
    float* reserved_space,
    float* reserved_activations,
    uint8_t* reserved_mask,
    void* lt_workspace,
    float p);

template int mlp_bp<float>(
    float* X,
    float* Y,
    int input_features,
    int batch_size,
    float** WPtr,
    float** BPtr,
    int num_layers,
    int* output_features,
    float* dY,
    float* reserved_space,
    float* reserved_activations,
    uint8_t* reserved_mask,
    float* work_space,
    float* dX,
    float** dwPtr,
    float** dbPtr,
    bool requires_grad,
    void* lt_workspace,
    float p,
    bool recompute);

template int mlp_bp_input_only<float>(
    float* X,
    float* Y,
    int input_features,
    int batch_size,
    float** WPtr,
    int num_layers,
    int* output_features,
    float* dY,
    float* reserved_space,
    float* reserved_activations,
    uint8_t* reserved_mask,
    float* work_space,
    float* dX,
    bool requires_grad,
    void* lt_workspace,
    float p);

template int mlp_fp<at::Half>(
    at::Half* X,
    int input_features,
    int batch_size,
    at::Half** WPtr,
    at::Half** BPtr,
    int num_layers,
    int* output_features,
    at::Half* Y,
    at::Half* reserved_space,
    at::Half* reserved_activations,
    uint8_t* reserved_mask,
    void* lt_workspace,
    float p);

template int mlp_bp<at::Half>(
    at::Half* X,
    at::Half* Y,
    int input_features,
    int batch_size,
    at::Half** WPtr,
    at::Half** BPtr,
    int num_layers,
    int* output_features,
    at::Half* dY,
    at::Half* reserved_space,
    at::Half* reserved_activations,
    uint8_t* reserved_mask,
    at::Half* work_space,
    at::Half* dX,
    at::Half** dwPtr,
    at::Half** dbPtr,
    bool requires_grad,
    void* lt_workspace,
    float p,
    bool recompute);

template int mlp_bp_input_only<at::Half>(
    at::Half* X,
    at::Half* Y,
    int input_features,
    int batch_size,
    at::Half** WPtr,
    int num_layers,
    int* output_features,
    at::Half* dY,
    at::Half* reserved_space,
    at::Half* reserved_activations,
    uint8_t* reserved_mask,
    at::Half* work_space,
    at::Half* dX,
    bool requires_grad,
    void* lt_workspace,
    float p);

template int mlp_fp<double>(
    double* X,
    int input_features,
    int batch_size,
    double** WPtr,
    double** BPtr,
    int num_layers,
    int* output_features,
    double* Y,
    double* reserved_space,
    double* reserved_activations,
    uint8_t* reserved_mask,
    void* lt_workspace,
    float p);

template int mlp_bp<double>(
    double* X,
    double* Y,
    int input_features,
    int batch_size,
    double** WPtr,
    double** Btr,
    int num_layers,
    int* output_features,
    double* dY,
    double* reserved_space,
    double* reserved_activations,
    uint8_t* reserved_mask,
    double* work_space,
    double* dX,
    double** dwPtr,
    double** dbPtr,
    bool requires_grad,
    void* lt_workspace,
    float p,
    bool recompute);

template int mlp_bp_input_only<double>(
    double* X,
    double* Y,
    int input_features,
    int batch_size,
    double** WPtr,
    int num_layers,
    int* output_features,
    double* dY,
    double* reserved_space,
    double* reserved_activations,
    uint8_t* reserved_mask,
    double* work_space,
    double* dX,
    bool requires_grad,
    void* lt_workspace,
    float p);




template size_t get_mlp_bp_workspace_in_bytes<float>(
    int batch_size,
    int num_layers,
    const int* output_features);
template size_t get_mlp_bp_workspace_in_bytes<at::Half>(
    int batch_size,
    int num_layers,
    const int* output_features);
template size_t get_mlp_bp_workspace_in_bytes<double>(
    int batch_size,
    int num_layers,
    const int* output_features);
