#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <torch/torch.h>
#include <cmath>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <hipblaslt.h>

// Keep Sigmoid in float only. When using half, cast to float before calling.
__device__ __inline__ float sigmoid(float a) {
  float retf = 1.f / (1.f + expf(-a));
  return (retf);
}

// Sigmoid. Assume input X is [features x batch size], column major.
// Safe to call in-place.
template <typename T>
__global__ void Sigmoid_fprop(T *X, uint batch_size, uint features) {
  T r_x[ILP];
  if(is_aligned(X) && features % ILP ==0) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      load_store(r_x, X, 0 , tid);
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        r_x[ii] = sigmoid(static_cast<float>(r_x[ii]));
      }
      load_store(X, r_x, tid , 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          r_x[ii] = X[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        r_x[ii] = sigmoid(static_cast<float>(r_x[ii]));
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          X[idx] = r_x[ii];
        }
      }
    }
  }
}

// Sigmoid. Assume input X is [features x batch size], column major.
// Safe to call in-place.
template <typename T>
__global__ void Sigmoid_bprop(T *dY, T *Y, uint batch_size, uint features, T *dX) {
  T r_dy[ILP];
  T r_y[ILP];
  if(is_aligned(dY) &&
     is_aligned(Y) &&
     is_aligned(dX) &&
     features % ILP ==0) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid*ILP < features * batch_size; tid += blockDim.x * gridDim.x) {
      load_store(r_dy, dY, 0 , tid);
      load_store(r_y, Y, 0 , tid);
#pragma unroll
      for(int ii=0;ii<ILP;ii++){
        float grad_out = r_dy[ii];
        float out = r_y[ii];
        float grad_i = out * ( 1.f - out) * grad_out;
        r_dy[ii] = grad_i;
      }
      load_store(dX, r_dy, tid, 0);
    }
  } else {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (; tid < features * batch_size; tid += ILP * blockDim.x * gridDim.x) {
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          r_dy[ii] = dY[idx];
          r_y[ii] = Y[idx];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        float grad_out = r_dy[ii];
        float out = r_y[ii];
        float grad_i = out * ( 1.f - out) * grad_out;
        r_dy[ii] = grad_i;
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++) {
        int idx = tid + ii * blockDim.x * gridDim.x;
        if(idx < features * batch_size) {
          dX[idx] = r_dy[ii];
        }
      }
    }
  }
}

// FP64 Wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    double* A,
    int lda,
    double* B,
    int ldb,
    const float* beta,
    double* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_64F,
      lda,
      B,
      HIP_R_64F,
      ldb,
      beta,
      C,
      HIP_R_64F,
      ldc,
      HIP_R_64F,
      HIPBLAS_GEMM_DEFAULT);
}

// FP32 Wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    float* A,
    int lda,
    float* B,
    int ldb,
    const float* beta,
    float* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_32F,
      lda,
      B,
      HIP_R_32F,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      HIPBLAS_GEMM_DEFAULT);
}

// FP16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float* beta,
    at::Half* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      beta,
      C,
      HIP_R_16F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}


#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600


int gemm_bias_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BIAS;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS)
    goto CLEANUP;


  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}







int gemm_bias_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    double* A,
    int lda,
    double* B,
    int ldb,
    const float *beta, /* host pointer */
    double* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bias) {
  return 1;
}

int gemm_bias_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    float *A,
    int lda,
    float *B,
    int ldb,
    const float *beta, /* host pointer */
    float *C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BIAS;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_32F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }

  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          &heuristicResult.algo,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}


int gemm_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    void* bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      printf("Fail 1");
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BGRADB;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("Fail 2");
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("Can't find workspace.\n");
    goto CLEANUP;
  }

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("can't find algorithm\n");
    goto CLEANUP;
  }

  if (returnedResults == 0) {
    printf("HIPBLAS_STATUS_NOT_SUPPORTED");
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}







int gemm_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    double* A,
    int lda,
    double* B,
    int ldb,
    const float *beta, /* host pointer */
    double* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    void* bgrad) {
  return 1;
}

int gemm_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    float *A,
    int lda,
    float *B,
    int ldb,
    const float *beta, /* host pointer */
    float *C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    void* bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BGRADB;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_32F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }

  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          &heuristicResult.algo,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}


#endif

template <typename T>
int linear_bias_forward_cuda(at::Tensor input, T *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    const float beta_one       = 1.0;
    int status = 1;
    status = gemm_bias_lt(
        (hipblasLtHandle_t)handle,
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        out_features,
        batch_size,
        in_features,
        &alpha, /* host pointer */
        weight,
        in_features,
        input.data_ptr<T>(),
        in_features,
        &beta_zero, /* host pointer */
        output.data_ptr<T>(),
        out_features,
        lt_workspace,
        1 << 22,
        stream,
        true,
        static_cast<const void*>(bias.data_ptr<T>()));

    if (status != 0){
//         printf("GEMM BIAS LT not available. Backoff to GEMM and manual bias.");
        return 1;
    }
    return status;
}


template <typename T>
int linear_bias_forward_sigmoid_cuda(at::Tensor input, T *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    const float beta_one       = 1.0;
    int status = 1;
    status = gemm_bias_lt(
        (hipblasLtHandle_t)handle,
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        out_features,
        batch_size,
        in_features,
        &alpha, /* host pointer */
        weight,
        in_features,
        input.data_ptr<T>(),
        in_features,
        &beta_zero, /* host pointer */
        output.data_ptr<T>(),
        out_features,
        lt_workspace,
        1 << 22,
        stream,
        true,
        static_cast<const void*>(bias.data_ptr<T>()));

    if (status != 0){
//         printf("GEMM BIAS LT not available. Backoff to GEMM and manual bias.");
        return 1;
    }
    return status;
}


template <typename T>
int linear_bias_backward_cuda(bool compute_grad_input, T *input, T *weight, T *d_output, int in_features, int batch_size, int out_features, T *d_weight, T *d_bias, T *d_input,  void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    int status = 1;

    status = gemm_bgradb_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_T,
    in_features,
    out_features,
    batch_size,
    &alpha, /* host pointer */
    input,
    in_features,
    d_output,
    out_features,
    &beta_zero, /* host pointer */
    d_weight,
    in_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<void*>(d_bias));

    if (status != 0){
        return 1;
    }

    if (compute_grad_input)
        status = gemm_bias(
          handle,
          HIPBLAS_OP_N,
          HIPBLAS_OP_N,
          in_features,
          batch_size,
          out_features,
          &alpha,
          weight,
          in_features,
          d_output,
          out_features,
          &beta_zero,
          d_input,
          in_features);

    return status;

}



template <typename T>
int linear_bias_backward_input_only_cuda(T *input, T *weight, T *d_output, int in_features, int batch_size, int out_features,
    T *d_input,  void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    int status = 1;

    status = gemm_bias(
      handle,
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      in_features,
      batch_size,
      out_features,
      &alpha,
      weight,
      in_features,
      d_output,
      out_features,
      &beta_zero,
      d_input,
      in_features);

    return status;

}

// FORWARD
template int linear_bias_forward_cuda<at::Half>(at::Tensor input, at::Half *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);

template int linear_bias_forward_cuda<float>(at::Tensor input, float *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);

template int linear_bias_forward_cuda<double>(at::Tensor input, double *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);

// FORWARD with sigmoid
template int linear_bias_sigmoid_forward_cuda<at::Half>(at::Tensor input, at::Half *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);

template int linear_bias_sigmoid_forward_cuda<float>(at::Tensor input, float *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);

template int linear_bias_sigmoid_forward_cuda<double>(at::Tensor input, double *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);

// BACKWARD
template int linear_bias_backward_cuda<at::Half>(bool compute_grad_input, at::Half *input, at::Half *weight, at::Half *d_output, int in_features, int batch_size, int out_features, at::Half *d_weight, at::Half *d_bias, at::Half *d_input,  void *lt_workspace) ;

template int linear_bias_backward_cuda<float>(bool compute_grad_input, float *input, float *weight, float *d_output, int in_features, int batch_size, int out_features, float *d_weight, float *d_bias, float *d_input,  void *lt_workspace) ;

template int linear_bias_backward_cuda<double>(bool compute_grad_input, double *input, double *weight, double *d_output, int in_features, int batch_size, int out_features, double *d_weight, double *d_bias, double *d_input,  void *lt_workspace) ;

// BACKWARD with sigmoid
template int linear_bias_sigmoid_backward_cuda<at::Half>(bool compute_grad_input, at::Half *input, at::Half *weight, at::Half *d_output, int in_features, int batch_size, int out_features, at::Half *d_weight, at::Half *d_bias, at::Half *d_input,  void *lt_workspace) ;

template int linear_bias_sigmoid_backward_cuda<float>(bool compute_grad_input, float *input, float *weight, float *d_output, int in_features, int batch_size, int out_features, float *d_weight, float *d_bias, float *d_input,  void *lt_workspace) ;

template int linear_bias_sigmoid_backward_cuda<double>(bool compute_grad_input, double *input, double *weight, double *d_output, int in_features, int batch_size, int out_features, double *d_weight, double *d_bias, double *d_input,  void *lt_workspace) ;

// BACKWARD input only (no weight computation)
template int linear_bias_backward_input_only_cuda<at::Half>(at::Half *input, at::Half *weight, at::Half *d_output, int in_features, int batch_size, int out_features, at::Half *d_input,  void *lt_workspace) ;

template int linear_bias_backward_input_only_cuda<float>(float *input, float *weight, float *d_output, int in_features, int batch_size, int out_features, float *d_input,  void *lt_workspace) ;

template int linear_bias_backward_input_only_cuda<double>(double *input, double *weight, double *d_output, int in_features, int batch_size, int out_features, double *d_input,  void *lt_workspace) ;
